#include "hip/hip_runtime.h"
// Each thread will output no more than one row to tempOutput_global. However
// each row may span multiple shared memory segments (due to spanning 
// multiple threads). Therefore we need to store in sharedmem at least two
// values per thread. These will be reduced down to one value per thread on 
// output.
extern "C" __global__ __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM)
void SPMXV_NAME(const uint* rowIndices_global, const uint* colIndices_global,
	 const MemType* sparseValues_global, uint numGroups,
	 ComputeType* tempOutput_global) {
	
	uint tid = threadIdx.x;
	uint lane = (WARP_SIZE - 1) & tid;
	uint warp = tid / WARP_SIZE;
	uint block = blockIdx.x;
	uint gid = NUM_WARPS * block + warp;

	// Shared memory index. Each threads needs two slots (64 per warp). For 
	// complex precision types, four slots are needed.
#ifdef USE_COMPLEX
	uint sharedOffset = 4 * WARP_SIZE * warp;
#else
	uint sharedOffset = 2 * WARP_SIZE * warp;
#endif
	uint sharedX = sharedOffset + lane;
	uint sharedY = sharedX + WARP_SIZE;

	// Load the row indices for each warp.
	if(tid < NUM_WARPS)
		sharedArray[tid] = rowIndices_global[min(numGroups - 1, gid + tid)];
	__syncthreads();
	
	uint rowIndex = sharedArray[warp];
	__syncthreads();
	
	// Break out of the kernel if the group is out of range
	if(gid >= numGroups) return;

	// offset0 is the offset of the first value of the current thread in 
	// colIndices/sparseValues. Add WARP_SIZE for each subsequent value in the 
	// thread.
	uint offset0 = WARP_SIZE * VALUES_PER_THREAD * gid + lane;
	
	// Load the column indices and sparse matrix values from global memory.
	// These are packed into the first four column indices for each thread.	
	uint colIndices[4];
	colIndices[0] = colIndices_global[offset0];
	colIndices[1] = colIndices_global[offset0 + WARP_SIZE];
	colIndices[2] = colIndices_global[offset0 + 2 * WARP_SIZE];
	colIndices[3] = colIndices_global[offset0 + 3 * WARP_SIZE];
				
	// Extract the offsets to execute the segmented scan.		
	uint scanOffset = (colIndices[0]>> 25) + sharedOffset;
	uint deltaPairX = colIndices[1]>> 26;
	uint deltaPairY = colIndices[2]>> 25;
	uint rowSumIndex = (colIndices[3]>> 25) + sharedOffset;
	
	// Although products may be up to 20 elements, it is not treated like an
	// actual array, taking that much space. The register usage of this kernel
	// should be low because, depending on how nvcc re-orders instructions to
	// increase ILP, only the last few members of products need be accessed.
	ComputeType products[VALUES_PER_THREAD];
	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {

		// Load the column index for this thread value. The first four have
		// already been loaded.
		uint offset = offset0 + WARP_SIZE * i;
		uint colIndex;
		if(i < 4) colIndex = colIndices[i];
		else colIndex = colIndices_global[offset];
		
		// Load the matrix value and up-convert into matrixValue.
		ComputeType matrixValue = ConvertUp(sparseValues_global[offset]);
		
		// Fetch the texture and up-convert into vectorValue.
		ComputeType vectorValue = FromTexture(
			tex1Dfetch(xVec_texture, 0x003fffff & colIndex));
		
		uint startFlag = FirstThreadRow & colIndex;
		uint endFlag = LastThreadRow & colIndex;

		ComputeType prev = startFlag ? products[i - 1] : Zero;

		// if(startFlag) prev = products[i - 1];
		products[i] = Add(Mul(matrixValue, vectorValue), prev);
		
		SetShared(scanOffset, products[i]);
		scanOffset += 0 != endFlag;
	}
	
	// Perform the segmented scan. Because this scan processes 64 values and
	// our warp is only 32 threads, each thread processes two slots separated by
	// 32 slots.
	ComputeType valueX = GetShared(sharedX);
	ComputeType valueY = GetShared(sharedY);
	
	// For all offsets < WARP_SIZE, we can handle the left and right halves of
	// sharedArray simultaneously, without intervening __syncthreads().
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		
		bool predX = offset <= deltaPairX;
		bool predY = offset <= deltaPairY;

		// Avoid putting multiple statements in a branch, because nvcc will
		// generate a BRA.U instruction rather than simple predication.
		if(predX) valueX = Add(valueX, GetShared(sharedX - offset));
		SetShared(sharedX, valueX);
		
		if(predY) valueY = Add(valueY, GetShared(sharedY - offset));
		SetShared(sharedY, valueY);
	}
	
	// For offset = WARP_SIZE, only handle the right half.
	bool predY = WARP_SIZE <= deltaPairY;
	if(predY) {
		valueY = Add(valueY, GetShared(sharedY - WARP_SIZE));
		SetShared(sharedY, valueY);
	}

	// Write the final row sums to tempOutput_gloabl	
	if(SerializeFlag & colIndices[1])
		// fetch the row sum from sharedArray
		tempOutput_global[rowIndex + lane] = GetShared(rowSumIndex);
}

#undef SPMXV_NAME
#undef VALUES_PER_THREAD
