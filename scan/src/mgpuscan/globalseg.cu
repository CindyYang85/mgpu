#include "hip/hip_runtime.h"

#define NUM_THREADS 256
#define NUM_WARPS (NUM_THREADS / WARP_SIZE)
#define LOG_NUM_WARPS 3

#define BLOCKS_PER_SM 4

#define VALUES_PER_THREAD 8
#define VALUES_PER_WARP (WARP_SIZE * VALUES_PER_THREAD)
#define NUM_VALUES (NUM_THREADS * VALUES_PER_THREAD)


////////////////////////////////////////////////////////////////////////////////
// UPSWEEP PASS. Find the sum of all values in the last segment in each block.
// When the first head flag in the block is encountered, write out the sum to 
// that point and return. We only need to reduce the last segment to feed sums
// up to the reduction pass.

DEVICE int Reduce(uint tid, int x, int code, int init) {

	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	const int ScanStride = WARP_SIZE + WARP_SIZE / 2 + 1;
	const int ScanSize = NUM_WARPS * ScanStride;
	__shared__ volatile int reduction_shared[ScanSize];
	__shared__ volatile int totals_shared[NUM_WARPS + NUM_WARPS / 2];

	volatile int* s = reduction_shared + ScanStride * warp + lane + 
		WARP_SIZE / 2;
	s[-16] = init;
	s[0] = x;

	// Run intra-warp max reduction.
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(0 == code) x += s[-offset];
		else if(1 == code) x = max(x, s[-offset]);
		s[0] = x;
	}

	// Synchronize to make all the totals available to the reduction code.
	__syncthreads();
	if(tid < NUM_WARPS) {
		// Grab the block total for the tid'th block. This is the last element
		// in the block's scanned sequence. This operation avoids bank 
		// conflicts.
		x = reduction_shared[ScanStride * tid + WARP_SIZE / 2 +
			WARP_SIZE - 1];

		totals_shared[tid] = init;
		volatile int* s = totals_shared + NUM_WARPS / 2 + tid;
		s[0] = x;
		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			int offset = 1<< i;
			if(0 == code) x += s[-offset];
			else if(1 == code) x = max(x, s[-offset]);
			s[0] = x;
		}

		if(NUM_WARPS - 1 == tid) totals_shared[0] = x;
	}

	// Synchronize to make the block scan available to all warps.
	__syncthreads();

	return totals_shared[0];
}

extern "C" __global__ __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM)
void SegScanUpsweepFlag(const uint* valuesIn_global, uint* blockLast_global,
	int* headFlagPos_global, const int2* rangePairs_global) {

	uint tid = threadIdx.x;
	uint block = blockIdx.x;

	int2 range = rangePairs_global[block];
	
	// Start at the last tile (NUM_VALUES before the end iterator). Because
	// upsweep isn't executed for the last block, we don't have to worry about
	// the ending edge case.
	int current = range.y - NUM_VALUES;

	uint threadSum = 0;
	int segmentStart = -1;

	while(current >= range.x) {

		uint packed[VALUES_PER_THREAD];
	
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) 
			packed[i] = valuesIn_global[current + tid + i * NUM_THREADS];
		

		// Find the index of the latest value loaded with a head flag set.
		int lastHeadFlagPos = -1;

		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint flag = 0x80000000 & packed[i];
			if(flag) lastHeadFlagPos = i;
		}
		if(-1 != lastHeadFlagPos)
			lastHeadFlagPos = tid + lastHeadFlagPos * NUM_THREADS;

		segmentStart = Reduce(tid, lastHeadFlagPos, 1, -1);

		// Make a second pass and sum all the values that appear at or after
		// segmentStart.

		// Add if tid + i * NUM_THREADS >= segmentStart.
		// Subtract tid from both sides to simplify expression.
		int cmp = segmentStart - tid;
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint value = 0x7fffffff & packed[i];
			if(i * NUM_THREADS >= cmp)
				threadSum += value;
		}
		if(-1 != segmentStart) break;

		current -= NUM_VALUES;
	}

	// We've either hit the head flag or run out of values. Do a horizontal sum
	// of the thread values and store to global memory.
	uint total = (uint)Reduce(tid, (int)threadSum, 0, 0);

	if(0 == tid) {
		blockLast_global[block] = total;
		int headFlag = -1 != segmentStart;
		if(-1 != segmentStart) segmentStart += current;
		headFlagPos_global[block] = headFlag;
	}
}

extern "C" __global__ __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM)
void SegScanUpsweepKeys(const uint* valuesIn_global, const uint* keysIn_global,
	uint* blockLast_global, const int2* rangePairs_global) {


}


////////////////////////////////////////////////////////////////////////////////
// REDUCTION PASS. 

extern "C" __global__ __launch_bounds__(NUM_THREADS, 1)
void SegScanReduction(const uint* headFlags_global, uint* blockLast_global,
	uint numBlocks) {

	uint tid = threadIdx.x;
	uint lane = (WARP_SIZE - 1) & tid;
	uint warp = tid / WARP_SIZE;
	
	// Load the head flag and last segment counts for each thread. These map
	// to blocks in the upsweep/downsweep passes.
	uint flag = 0;
	uint x = 0;
	if(tid < numBlocks) {
		flag = headFlags_global[tid];
		x = blockLast_global[tid];
	}

	// Get the start flags for each thread in the warp.
	uint flags = __ballot(flag);

	// Mask out the bits at or above the current lane.
	uint mask = bfi(0, 0xffffffff, 0, lane);
	uint flagsMasked = flags & mask;

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	int preceding = 31 - __clz(flagsMasked);
	uint distance = lane - preceding;

	__shared__ volatile uint shared[NUM_WARPS * (WARP_SIZE + 1)];
	__shared__ volatile uint blockShared[2 * NUM_WARPS];
	volatile uint* warpShared = shared + warp * (WARP_SIZE + 1) + 1;
	volatile uint* threadShared = warpShared + lane;

	// Run an inclusive scan for each warp. This does not require any special 
	// treatment of segment edges, as we have only one value per thread.
	threadShared[-1] = 0;
	threadShared[0] = x;
	uint sum = x;
	uint first = warpShared[preceding];

	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		int offset = 1<< i;
		if(distance > offset)
			sum += threadShared[-offset];
		threadShared[0] = sum; 
	}
	sum += first;

	uint last = flag ? x : sum;

	// sum now holds the inclusive scan for the part of the segment within the
	// warp. Run a multiscan by having each warp store its flags value to
	// shared memory.
	if(WARP_SIZE - 1 == lane) {
		blockShared[warp] = flags;
		blockShared[NUM_WARPS + warp] = last;
	}

	__syncthreads();
	if(tid < NUM_WARPS) {

		// Load the inclusive sums for the last value in each warp and the head
		// flags for each warp.
		uint flag = blockShared[tid];
		uint x = blockShared[NUM_WARPS + tid];
		uint flags = __ballot(flag) & mask;

		int preceding = 31 - __clz(flags);
		uint distance = tid - preceding;
		
		volatile uint* s = blockShared + NUM_WARPS + tid;
		s[-NUM_WARPS] = 0;

		uint sum = x;
		uint first = blockShared[NUM_WARPS + preceding];

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			int offset = 1<< i;
			if(distance > offset) sum += s[-offset];
			s[0] = sum;
		}
		
		// Add preceding and subtract x to get an exclusive sum.
		sum += first - x;

		blockShared[tid] = sum;
	}

	__syncthreads();

	uint blockScan = blockShared[warp];

	// Add blockScan if the warp doesn't hasn't encountered a head flag yet.
	if(!flagsMasked) sum += blockScan;
	sum -= x;

	if(tid < numBlocks)
		blockLast_global[tid] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// INTER-WARP REDUCTION 
// Calculate the length of the last segment in the last lane in each warp.

DEVICE uint BlockScan(uint tid, uint warp, uint lane, uint last,
	uint warpFlags, uint mask) {

	__shared__ volatile uint blockShared[3 * NUM_WARPS];
	if(WARP_SIZE - 1 == lane) {
		blockShared[NUM_WARPS + warp] = last;
		blockShared[2 * NUM_WARPS + warp] = warpFlags;
	}
	__syncthreads();

	if(tid < NUM_WARPS) {
		// Pull out the sum and flags for each warp.
		volatile uint* s = blockShared + NUM_WARPS + lane;
		uint warpLast = s[0];
		uint flag = s[NUM_WARPS];
		s[-NUM_WARPS] = 0;

		uint blockFlags = __ballot(flag);

		// Mask out the bits at or above the current warp.
		blockFlags &= mask;

		// Find the distance from the current warp to the warp at the start of 
		// this segment.
		int preceding = 31 - __clz(blockFlags);
		uint distance = lane - preceding;
		

		// INTER-WARP reduction
		uint warpSum = warpLast;
		uint warpFirst = blockShared[NUM_WARPS + preceding];

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			uint offset = 1<< i;
			if(distance > offset) warpSum += s[-offset];
			if(i < LOG_NUM_WARPS - 1) s[0] = warpSum;
		}
		// Subtract warpLast to make exclusive and add first to grab the
		// fragment sum of the preceding warp.
		warpSum += warpFirst - warpLast;

		// Store warpSum back into shared memory. This is added to all the
		// lane sums and those are added into all the threads in the first 
		// segment of each lane.
		blockShared[lane] = warpSum;
	}
	__syncthreads();

	return blockShared[warp];
}


DEVICE void SegScanDownsweep(uint tid, uint lane, uint warp, 
	const uint x[VALUES_PER_THREAD], const uint flags[VALUES_PER_THREAD],
	uint scan[VALUES_PER_THREAD], volatile uint* warpShared, 
	volatile uint* threadShared, bool inclusive, 
	volatile uint* blockOffset_shared) {

	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP PASS
	// Add sum to all the values in the continuing segment (that is, before the
	// first start flag) in this thread.

	uint blockOffset = 0;
	if(!tid) blockOffset = *blockOffset_shared;
	uint last = blockOffset;

	// Compute the exclusive scan into scan. These values are then added to the
	// final thread offsets after the inter-warp multiscan pattern.
	uint hasHeadFlag = 0;

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		if(flags[i]) last = 0;
		hasHeadFlag |= flags[i];

		scan[i] = last;
		if(inclusive) scan[i] += x[i];
		last += x[i];
	}


	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP SEGMENT PASS
	// Run a ballot and clz to find the lane containing the start value for the
	// segment that begins this thread.

	uint warpFlags = __ballot(hasHeadFlag);

	// Mask out the bits at or above the current thread.
	uint mask = bfi(0, 0xffffffff, 0, lane);
	uint warpFlagsMask = warpFlags & mask;

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	int preceding = 31 - __clz(warpFlagsMask);
	uint distance = lane - preceding;


	////////////////////////////////////////////////////////////////////////////
	// REDUCTION PASS
	// Run a prefix sum scan over last to compute for each lane the sum of all
	// values in the segmented preceding the current lane, up to that point.
	// This is added back into the thread-local exclusive scan for the continued
	// segment in each thread.
	
	volatile uint* shifted = threadShared + 1;
	shifted[-1] = 0;
	shifted[0] = last;
	uint sum = last;
	uint first = warpShared[1 + preceding];

	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(distance > offset) sum += shifted[-offset];
		if(i < LOG_WARP_SIZE - 1) shifted[0] = sum;
	}
	// Subtract last to make exclusive and add first to grab the fragment
	// sum of the preceding thread.
	sum += first - last;

	// Call BlockScan for inter-warp scan on the reductions of the last
	// segment in each warp.
	uint lastSegLength = last;
	if(!hasHeadFlag) lastSegLength += sum;

	uint blockScan = BlockScan(tid, warp, lane, lastSegLength, warpFlags, 
		mask);
	if(!warpFlagsMask) sum += blockScan;

	if(NUM_THREADS - 1 == tid) {
		if(!hasHeadFlag) last += sum;
		*blockOffset_shared = last;
	}

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		if(flags[i]) sum = 0;
		scan[i] += sum;
	}
}


extern "C" __global__ __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM)
void SegScanDownsweepFlag(const uint* valuesIn_global, uint* valuesOut_global,
	const uint* start_global, const int2* rangePairs_global, int count,
	int inclusive) {

	uint tid = threadIdx.x;
	uint lane = (WARP_SIZE - 1) & tid;
	uint warp = tid / WARP_SIZE;
	uint block = blockIdx.x;
	uint index = VALUES_PER_WARP * warp + lane;

	int2 range = rangePairs_global[block];

	const int Size = NUM_WARPS * VALUES_PER_THREAD * (WARP_SIZE + 1);
	__shared__ volatile uint shared[Size];
	__shared__ volatile uint blockOffset_shared;

	// Use a stride of 33 slots per warp per value to allow conflict-free
	// transposes from strided to thread order.
	volatile uint* warpShared = shared + 
		warp * VALUES_PER_THREAD * (WARP_SIZE + 1);
	volatile uint* threadShared = warpShared + lane;
	
	// Transpose values into thread order.
	uint offset = VALUES_PER_THREAD * lane;
	offset += offset / WARP_SIZE;

	int lastOffset = ~(NUM_VALUES - 1) & count;


	if(!tid) blockOffset_shared = start_global[block];


	while(range.x < range.y) {
		// Load values into packed.
		uint x[VALUES_PER_THREAD];
		uint flags[VALUES_PER_THREAD];

			#pragma unroll
			for(int i = 0; i < VALUES_PER_THREAD; ++i) {
				uint source = range.x + index + i * WARP_SIZE;
				uint x = valuesIn_global[source];
				threadShared[i * (WARP_SIZE + 1)] = x;
			}

		// Transpose into thread order 
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint packed = warpShared[offset + i];
			x[i] = 0x7fffffff & packed;
			flags[i] = 0x80000000 & packed;
		}
	


		uint scan[VALUES_PER_THREAD];
		SegScanDownsweep(tid, lane, warp, x, flags, scan, warpShared, 
			threadShared, inclusive, &blockOffset_shared);

		// Transpose 
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i)
			warpShared[offset + i] = scan[i];

		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint target = range.x + index + i * WARP_SIZE;
			valuesOut_global[target] = threadShared[i * (WARP_SIZE + 1)];
		}
		



		// Load values
	/*	if(range.x >= lastOffset) {
			// Use conditional loads.
			#pragma unroll
			for(int i = 0; i < VALUES_PER_THREAD; ++i) {
				uint x = 0;
				uint source = range.x + index + i * WARP_SIZE;
				if(source < count) x = valuesIn_global[source];
				threadShared[i * (WARP_SIZE + 1)] = x;
			}
		} else {*/
		//}

		__syncthreads();





		// Store values
		/*if(range.x >= lastOffset) {
			// Use conditional loads.
			#pragma unroll
			for(int i = 0; i < VALUES_PER_THREAD; ++i) {
				uint x = 0;
				uint source = range.x + index + i * WARP_SIZE;
				if(source < count) x = valuesIn_global[source];
				threadShared[i * (WARP_SIZE + 1)] = x;
			}
		} else {*/
	
	//	}

		range.x += NUM_VALUES;
	}
}

extern "C" __global__ __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM)
void SegScanDownsweepKeys(const uint* valuesIn_global, uint* valuesOut_global,
	const uint* start_global, const int2* rangePairs_global, int count,
	uint init, bool inclusive) {

}

#undef NUM_THREADS
#undef NUM_WARPS
#undef LOG_NUM_WARPS
#undef BLOCKS_PER_SM
#undef VALUES_PER_THREAD
#undef VALUES_PER_WARP
#undef NUM_VALUES
