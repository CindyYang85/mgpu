#include "hip/hip_runtime.h"

#define NUM_THREADS 1024
#define NUM_WARPS (NUM_THREADS / WARP_SIZE)
#define LOG_NUM_WARPS 5

#define BLOCKS_PER_SM 1

#define VALUES_PER_THREAD 4
#define VALUES_PER_WARP (WARP_SIZE * VALUES_PER_THREAD)
#define NUM_VALUES (NUM_THREADS * VALUES_PER_THREAD)


////////////////////////////////////////////////////////////////////////////////
// Multiscan utility function. Used in the first and third passes of the
// global scan function. Returns the inclusive scan of the arguments in .x and
// the sum of all arguments in .y.

// Each warp is passed a pointer to its own contiguous area of shared memory.
// There must be at least 48 slots of memory. They should also be aligned so
// that the difference between the start of consecutive warps differ by an 
// interval that is relatively prime to 32 (any odd number will do).

DEVICE uint2 Multiscan(uint tid, uint x, volatile uint* warpShared) {

	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	volatile uint* s = warpShared + lane + WARP_SIZE / 2;
	warpShared[lane] = 0;
	s[0] = x;

	// Run inclusive scan on each warp's data.
	uint sum = x;	
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		sum += s[-offset];
		if(i < LOG_WARP_SIZE - 1) s[0] = sum;
	}

	__shared__ volatile uint totals_shared[2 * NUM_WARPS];
	if(WARP_SIZE - 1 == lane) {
		totals_shared[NUM_WARPS + warp] = sum;
	}

	// Synchronize to make all the totals available to the reduction code.
	__syncthreads();
	if(tid < NUM_WARPS) {
		// Grab the block total for the tid'th block. This is the last element
		// in the block's scanned sequence. This operation avoids bank 
		// conflicts.
		uint total = totals_shared[NUM_WARPS + tid];
		totals_shared[tid] = 0;
		volatile uint* s = totals_shared + NUM_WARPS + tid;

		uint totalsSum = total;

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			int offset = 1<< i;
			totalsSum += s[-offset];
			s[0] = totalsSum;	
		}

		// Subtract total from totalsSum for an exclusive scan.
		totals_shared[tid] = totalsSum - total;
	}

	// Synchronize to make the block scan available to all warps.
	__syncthreads();

	// Add the block scan to the inclusive sum for the block.
	sum += totals_shared[warp];
	uint total = totals_shared[2 * NUM_WARPS - 1];
	return make_uint2(sum, total);
}

DEVICE uint2 Multiscan2(uint tid, uint x) {
	uint warp = tid / WARP_SIZE;
	const int WarpStride = WARP_SIZE + WARP_SIZE / 2;
	const int SharedSize = NUM_WARPS * WarpStride;
	__shared__ volatile uint shared[SharedSize];
	volatile uint* warpShared = shared + warp * WarpStride;
	return Multiscan(tid, x, warpShared);
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanUpsweep adds up all the values in elements_global within the 
// range given by blockCount and writes to blockTotals_global[blockIdx.x].

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanUpsweep(const uint* valuesIn_global, const uint2* range_global,
	uint* blockTotals_global) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	uint2 range = range_global[block];

	// Loop through all elements in the interval, adding up values.
	// There is no need to synchronize until we perform the multiscan.
	uint sum = 0;
	for(uint index = range.x + tid; index < range.y; index += 2 * NUM_THREADS)
		sum += valuesIn_global[index] + valuesIn_global[index + NUM_THREADS];

	// A full multiscan is unnecessary here - we really only need the total.
	// But this is easy and won't slow us down since this kernel is already
	// bandwidth limited.
	uint total = Multiscan2(tid, sum).y;

	if(!tid)
		blockTotals_global[block] = total;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanReduction performs an exclusive scan on the elements in 
// blockTotals_global and writes back in-place.

extern "C" __global__ void GlobalScanReduction(uint* blockTotals_global, 
	uint numBlocks) {

	uint tid = threadIdx.x;
	uint x = 0; 
	if(tid < numBlocks) x = blockTotals_global[tid];

	// Subtract the value from the inclusive scan for the exclusive scan.
	uint2 scan = Multiscan2(tid, x);
	if(tid < numBlocks) blockTotals_global[tid] = scan.x - x;

	// Have the first thread in the block set the scan total.
	if(!tid) blockTotals_global[numBlocks] = scan.y;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanDownsweep runs an exclusive scan on the same interval of data as in
// pass 1, and adds blockScan_global[blockIdx.x] to each of them, writing back
// out in-place.

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanDownsweep(const uint* valuesIn_global, uint* valuesOut_global,
	const uint* blockScan_global, const int2* range_global, int count, 
	int inclusive) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;
	uint index = VALUES_PER_WARP * warp + lane;

	uint blockScan = blockScan_global[block];
	int2 range = range_global[block];


	// Allocate 33 slots of shared memory per warp of data read. This allows
	// use to perform a conflict-free transpose from strided order to thread
	// order.
	const int Size = NUM_WARPS * VALUES_PER_THREAD * (WARP_SIZE + 1);
	__shared__ volatile uint shared[Size];

	// warpShared points to the start of the warp's data.
	volatile uint* warpShared = shared +
		warp * VALUES_PER_THREAD * (WARP_SIZE + 1);
	volatile uint* threadShared = warpShared + lane;	

	// Transpose values into thread order.
	uint offset = VALUES_PER_THREAD * lane;
	offset += offset / WARP_SIZE;

	while(range.x < range.y) {

		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint source = range.x + index + i * WARP_SIZE;
			uint x = valuesIn_global[source];

			threadShared[i * (WARP_SIZE + 1)] = x;
		}

		// Transpose into thread order by reading from transposeValues.
		// Compute the exclusive or inclusive scan of the thread values and 
		// their sum.
		uint scan[VALUES_PER_THREAD];
		uint sum = 0;
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = warpShared[offset + i];
			scan[i] = sum;
			if(inclusive) scan[i] += x;
			sum += x;
		}

		// Multiscan for each thread's scan offset within the block. Subtract
		// sum to make it an exclusive scan.
		uint2 localScan = Multiscan(tid, sum, warpShared);
		uint scanOffset = localScan.x + blockScan - sum;

		// Add the scan offset to each exclusive scan and put the values back
		// into the shared memory they came out of.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = scan[i] + scanOffset;
			warpShared[offset + i] = x;
		}

		// Store the scan back to global memory.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = threadShared[i * (WARP_SIZE + 1)];
			uint target = range.x + index + i * WARP_SIZE;
			valuesOut_global[target] = x;
		}

		// Grab the last element of totals_shared, which was set in Multiscan.
		// This is the total for all the values encountered in this pass.
		blockScan += localScan.y;

		range.x += NUM_VALUES;
	}
}

#undef NUM_THREADS
#undef NUM_WARPS
#undef LOG_NUM_WARPS
#undef BLOCKS_PER_SM
#undef VALUES_PER_THREAD
#undef VALUES_PER_WARP
#undef NUM_VALUES
