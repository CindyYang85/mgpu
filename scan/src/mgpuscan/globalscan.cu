#include "hip/hip_runtime.h"

#define NUM_THREADS 256
#define NUM_WARPS (NUM_THREADS / WARP_SIZE)
#define LOG_NUM_WARPS 3

#define BLOCKS_PER_SM 2

#define VALUES_PER_THREAD 16
#define VALUES_PER_WARP (WARP_SIZE * VALUES_PER_THREAD)
#define NUM_VALUES (NUM_THREADS * VALUES_PER_THREAD)


////////////////////////////////////////////////////////////////////////////////
// Multiscan utility function. Used in the first and third passes of the
// global scan function. Returns the inclusive scan of the arguments in .x and
// the sum of all arguments in .y.

// Each warp is passed a pointer to its own contiguous area of shared memory.
// There must be at least 48 slots of memory. They should also be aligned so
// that the difference between the start of consecutive warps differ by an 
// interval that is relatively prime to 32 (any odd number will do).
DEVICE uint2 Multiscan(uint tid, uint x, volatile uint* warpShared,
	int warpStride) {

	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;


	__shared__ volatile uint totals_shared[NUM_WARPS + NUM_WARPS / 2];

	volatile uint* s = reduction_shared + ScanStride * warp + lane + 
		WARP_SIZE / 2;
	s[-16] = 0;
	s[0] = x;

	// Run inclusive scan on each warp's data.
	uint sum = x;	
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		sum += s[-offset];
		s[0] = sum;
	}

	// Synchronize to make all the totals available to the reduction code.
	__syncthreads();
	if(tid < NUM_WARPS) {
		// Grab the block total for the tid'th block. This is the last element
		// in the block's scanned sequence. This operation avoids bank 
		// conflicts.
		uint total = reduction_shared[ScanStride * tid + WARP_SIZE / 2 +
			WARP_SIZE - 1];

		totals_shared[tid] = 0;
		volatile uint* s2 = totals_shared + NUM_WARPS / 2 + tid;
		uint totalsSum = total;
		s2[0] = total;

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			int offset = 1<< i;
			totalsSum += s2[-offset];
			s2[0] = totalsSum;	
		}

		// Subtract total from totalsSum for an exclusive scan.
		totals_shared[tid] = totalsSum - total;
	}

	// Synchronize to make the block scan available to all warps.
	__syncthreads();

	// Add the block scan to the inclusive sum for the block.
	sum += totals_shared[warp];
	uint total = totals_shared[NUM_WARPS + NUM_WARPS / 2 - 1];
	return make_uint2(sum, total);
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanUpsweep adds up all the values in elements_global within the 
// range given by blockCount and writes to blockTotals_global[blockIdx.x].

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanUpsweep(const uint* valuesIn_global, const uint2* range_global,
	uint* blockTotals_global) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	uint2 range = range_global[block];

	// Loop through all elements in the interval, adding up values.
	// There is no need to synchronize until we perform the multiscan.
	uint sum = 0;
	for(uint index = range.x + tid; index < range.y; index += NUM_THREADS)
		sum += valuesIn_global[index];

	// A full multiscan is unnecessary here - we really only need the total.
	// But this is easy and won't slow us down since this kernel is already
	// bandwidth limited.
	uint total = Multiscan(tid, sum).y;

	// The last scan element in the block is the total for all values summed
	// in this block.
	if(tid == NUM_THREADS - 1)
		blockTotals_global[block] = total;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanReduction performs an exclusive scan on the elements in 
// blockTotals_global and writes back in-place.

extern "C" __global__ void GlobalScanReduction(uint* blockTotals_global, 
	uint numBlocks) {

	uint tid = threadIdx.x;
	uint x = 0; 
	if(tid < numBlocks) x = blockTotals_global[tid];

	// Subtract the value from the inclusive scan for the exclusive scan.
	uint2 scan = Multiscan(tid, x);
	if(tid < numBlocks) blockTotals_global[tid] = scan.x - x;

	// Have the first thread in the block set the scan total.
	if(!tid) blockTotals_global[numBlocks] = scan.y;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanDownsweep runs an exclusive scan on the same interval of data as in
// pass 1, and adds blockScan_global[blockIdx.x] to each of them, writing back
// out in-place.

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanDownsweep(const uint* valuesIn_global, uint* valuesOut_global,
	const uint* blockScan_global, const int2* range_global, int count, 
	int inclusive) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	uint blockScan = blockScan_global[block];
	int2 range = range_global[block];


	// Allocate 33 slots of shared memory per warp of data read. This allows
	// use to perform a conflict-free transpose from strided order to thread
	// order.
	const int WarpStride = VALUES_PER_THREAD * (WARP_SIZE + 1);
	const int SharedSize = NUM_WARPS * WarpStride;

	__shared__ volatile uint shared[SharedSize];

	// warpValues points to the start of the warp's data.
	volatile uint* warpValues = shared + warp * WarpStride;
	volatile uint* threadValues = warpValues + lane;	

	// Have each warp read a consecutive block of memory. Because threads in a
	// warp are implicitly synchronized, we can "transpose" the terms into
	// thread-order without a __syncthreads().
	uint first = range.x + warp * (VALUES_PER_THREAD * WARP_SIZE) + lane;
	uint end = ROUND_UP(range.y, NUM_VALUES);

	uint valueOffset = lane * VALUES_PER_THREAD;
	volatile uint* transposeValues = warpValues + valueOffset + 
		valueOffset / WARP_SIZE;

	for(uint index = first; index < end; index += NUM_VALUES) {

		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint index2 = index + i * WARP_SIZE;
			uint value = 0;
			if(index2 < range.y) value = valuesIn_global[index2];

			threadValues[i * SHARED_STRIDE] = value;
		}

		// Transpose into thread order by reading from transposeValues.
		// Compute the exclusive or inclusive scan of the thread values and 
		// their sum.
		uint scan[VALUES_PER_THREAD];
		uint sum = 0;
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = transposeValues[i];
			scan[i] = sum;
			if(inclusive) scan[i] += x;
			sum += x;
		}

		// Multiscan for each thread's scan offset within the block. Subtract
		// sum to make it an exclusive scan.
		uint2 localScan = Multiscan(tid, sum);
		uint scanOffset = localScan.x + blockScan - sum;

		// Add the scan offset to each exclusive scan and put the values back
		// into the shared memory they came out of.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = scan[i] + scanOffset;
			transposeValues[i] = x;
		}

		// Store the scan back to global memory.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = threadValues[i * SHARED_STRIDE];
			uint index2 = index + i * WARP_SIZE;			
			if(index2 < range.y) valuesOut_global[index2] = x;
		}

		// Grab the last element of totals_shared, which was set in Multiscan.
		// This is the total for all the values encountered in this pass.
		blockScan += localScan.y;
	}
}

#undef NUM_THREADS
#undef NUM_WARPS
#undef LOG_NUM_WARPS
#undef BLOCKS_PER_SM
#undef VALUES_PER_THREAD
#undef VALUES_PER_WARP
#undef NUM_VALUES
