// Demonstration of segmented scan. 
// See http://www.moderngpu.com/sparse/segscan.html

#define WARP_SIZE 32
#define LOG_WARP_SIZE 5

#define NUM_THREADS 256
#define NUM_WARPS 8
#define LOG_NUM_WARPS 3

#define VALUES_PER_THREAD 8
#define VALUES_PER_WARP (WARP_SIZE * VALUES_PER_THREAD)

#define DEVICE extern "C" __device__ __forceinline__


#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

typedef unsigned int uint;

DEVICE uint bfi(uint x, uint y, uint bit, uint numBits) {
	uint ret;
	asm("bfi.b32 %0, %1, %2, %3, %4;" : 
		"=r"(ret) : "r"(y), "r"(x), "r"(bit), "r"(numBits));
	return ret;
}


////////////////////////////////////////////////////////////////////////////////
// Use ballot and clz to run a segmented scan over a single warp, with one value
// per thread.

extern "C" __global__ void SegScanWarp(const uint* dataIn_global,
	uint* dataOut_global) {

	uint tid = threadIdx.x;
	uint packed = dataIn_global[tid];

	// The start flag is in the high bit.
	uint flag = 0x80000000 & packed;

	// Get the start flags for each thread in the warp.
	uint flags = __ballot(flag);

	// Mask out the bits above the current thread.
	flags &= bfi(0, 0xffffffff, 0, tid + 1);

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	uint distance =  __clz(flags) + tid - 31;

	__shared__ volatile uint shared[WARP_SIZE];

	uint x = 0x7fffffff & packed;
	uint x2 = x;
	shared[tid] = x;

	// Perform the parallel scan. Note the conditional if(offset < distance)
	// replaces the ordinary scan conditional if(offset <= tid).
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(offset <= distance) x += shared[tid - offset];
		shared[tid] = x;
	}

	// Turn inclusive scan into exclusive scan.
	x -= x2;

	dataOut_global[tid] = x;
}


////////////////////////////////////////////////////////////////////////////////
// Use parallel scan to compute the ranges for a segmented scan over a warp with
// eight values per thread.

extern "C" __global__ void SegScanWarp8(const uint* dataIn_global,
	uint* dataOut_global) {

	uint tid = threadIdx.x;
	
	__shared__ volatile uint shared[VALUES_PER_THREAD * (WARP_SIZE + 1)];
	
	// Load packed values from global memory and scatter to shared memory. Use
	// a 33-slot stride between successive values in each thread to set us up
	// for a conflict-free strided order -> thread order transpose.

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = dataIn_global[i * WARP_SIZE + tid];
		shared[i * (WARP_SIZE + 1) + tid] = x;
	}

	uint offset = VALUES_PER_THREAD * tid;
	offset += offset / WARP_SIZE;
	uint packed[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i)
		packed[i] = shared[offset + i];


	////////////////////////////////////////////////////////////////////////////
	// UPSWEEP PASS
	// Run a sequential segmented scan for all values in the packed array. Find
	// the sum of all values in the thread's last segment. Additionally set
	// index to tid if any segments begin in this thread.
	
	uint last = 0;
	uint hasHeadFlag = 0;

	uint x[VALUES_PER_THREAD];
	uint flags[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		flags[i] = 0x80000000 & packed[i];
		x[i] = 0x7fffffff & packed[i];
		if(flags[i]) last = 0;
		hasHeadFlag |= flags[i];
		last += x[i];
	}


	////////////////////////////////////////////////////////////////////////////
	// SEGMENT PASS
	// Run a ballot and clz to find the thread containing the start value for
	// the segment that begins this thread.

	uint warpFlags = __ballot(hasHeadFlag);

	// Mask out the bits at or above the current thread.
	warpFlags &= bfi(0, 0xffffffff, 0, tid);

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	int preceding = 31 - __clz(warpFlags);
	uint distance = tid - preceding;


	////////////////////////////////////////////////////////////////////////////
	// REDUCTION PASS
	// Run a prefix sum scan over last to compute for each thread the sum of all
	// values in the segmented preceding the current thread, up to that point.
	// This is added back into the thread-local exclusive scan for the continued
	// segment in each thread.

	shared[tid] = 0;
	volatile uint* shifted = shared + tid + 1;
	
	shifted[0] = last;
	uint sum = last;
	uint first = shared[1 + preceding];

	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(distance > offset) sum += shifted[-offset];
		shifted[0] = sum;
	}
	// Subtract last to make exclusive and add first to grab the fragment sum of
	// the preceding thread.
	sum += first - last;


	////////////////////////////////////////////////////////////////////////////
	// DOWNSWEEP PASS
	// Add sum to all the values in the continuing segment (that is, before the
	// first start flag) in this thread.

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		if(flags[i]) sum = 0;
		shared[offset + i] = sum;
		sum += x[i];
	}

	// Store the values back to global memory.
	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = shared[i * (WARP_SIZE + 1) + tid];
		dataOut_global[i * WARP_SIZE + tid] = x;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Use a multiscan pattern to execute segmented scan over an entire block with
// 8 values per thread.

extern "C" __global__ __launch_bounds__(NUM_THREADS, 4) 
void SegScanBlock8(const uint* dataIn_global, uint* dataOut_global) {

	uint tid = threadIdx.x;
	uint lane = (WARP_SIZE - 1) & tid;
	uint warp = tid / WARP_SIZE;
	
	const int Size = NUM_WARPS * VALUES_PER_THREAD * (WARP_SIZE + 1);
	__shared__ volatile uint shared[Size];
	__shared__ volatile uint blockShared[3 * NUM_WARPS];

	// Use a stride of 33 slots per warp per value to allow conflict-free
	// transposes from strided to thread order.
	volatile uint* warpShared = shared + 
		warp * VALUES_PER_THREAD * (WARP_SIZE + 1);
	volatile uint* threadShared = warpShared + lane;
	

	////////////////////////////////////////////////////////////////////////////
	// Load packed values from global memory and scatter to shared memory. Use
	// a 33-slot stride between successive values in each thread to set us up
	// for a conflict-free strided order -> thread order transpose. Storing to
	// separate memory intervals allows use transpose without explicit
	// synchronization.

	uint index = VALUES_PER_WARP * warp + lane;

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = dataIn_global[index + i * WARP_SIZE];
		threadShared[i * (WARP_SIZE + 1)] = x;
	}

	uint offset = VALUES_PER_THREAD * lane;
	offset += offset / WARP_SIZE;
	uint packed[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i)
		packed[i] = warpShared[offset + i];


	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP UPSWEEP PASS
	// Run a sequential segmented scan for all values in the packed array. Find
	// the sum of all values in the thread's last segment. Additionally set
	// index to tid if any segments begin in this thread.
	
	uint last = 0;
	uint hasHeadFlag = 0;

	uint x[VALUES_PER_THREAD];
	uint flags[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		flags[i] = 0x80000000 & packed[i];
		x[i] = 0x7fffffff & packed[i];
		if(flags[i]) last = 0;
		hasHeadFlag |= flags[i];
		last += x[i];
	}


	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP SEGMENT PASS
	// Run a ballot and clz to find the lane containing the start value for
	// the segment that begins this thread.

	uint warpFlags = __ballot(hasHeadFlag);

	// Mask out the bits at or above the current thread.
	uint warpFlagsMask = warpFlags & bfi(0, 0xffffffff, 0, lane);

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	int preceding = 31 - __clz(warpFlagsMask);
	uint distance = lane - preceding;


	////////////////////////////////////////////////////////////////////////////
	// REDUCTION PASS
	// Run a prefix sum scan over last to compute for each lane the sum of all
	// values in the segmented preceding the current lane, up to that point.
	// This is added back into the thread-local exclusive scan for the continued
	// segment in each thread.
	
	volatile uint* shifted = threadShared + 1;
	shifted[-1] = 0;
	shifted[0] = last;
	uint sum = last;
	uint first = warpShared[1 + preceding];

	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(distance > offset) sum += shifted[-offset];
		shifted[0] = sum;
	}
	// Subtract last to make exclusive and add first to grab the fragment sum of
	// the preceding thread.
	sum += first - last;

	
	////////////////////////////////////////////////////////////////////////////
	// INTER-WARP REDUCTION 
	// Calculate the length of the last segment in the last lane in each warp.

	__syncthreads();
	if(WARP_SIZE - 1 == lane) {
		uint lastSegLength = last;
		if(!hasHeadFlag) lastSegLength += sum;

		blockShared[NUM_WARPS + warp] = lastSegLength;
		blockShared[2 * NUM_WARPS + warp] = warpFlags;
	}
	__syncthreads();
	

	if(tid < NUM_WARPS) {
		blockShared[tid] = 0;
		uint warpLast = blockShared[NUM_WARPS + tid];
		uint warpSegFlag = blockShared[2 * NUM_WARPS + tid];

		uint blockFlags = __ballot(warpSegFlag);

		// Mask out the bits at or above the current warp.
		blockFlags &= bfi(0, 0xffffffff, 0, tid);

		// Find the distance from the current warp to the warp at the start of 
		// this segment.
		int preceding = 31 - __clz(blockFlags);
		uint distance = tid - preceding;

		// INTER-WARP REDUCTION
		blockShared[tid] = 0;

		volatile uint* shifted = blockShared + NUM_WARPS + tid;
		uint warpSum = warpLast;
		uint warpFirst = blockShared[NUM_WARPS + preceding];

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			uint offset = 1<< i;
			if(distance > offset) warpSum += shifted[-offset];
			shifted[0] = warpSum;
		}
		// Subtract warpLast to make exclusive and add first to grab the
		// fragment sum of the preceding warp.
		warpSum += warpFirst - warpLast;

		// Store warpSum back into shared memory. This is added to all the
		// lane sums and those are added into all the threads in the first 
		// segment of each lane.
		blockShared[tid] = warpSum;
	}
	__syncthreads();


	// If there are no head flags before this thread in the warp, add the block
	// scan to sum.
	if(!warpFlagsMask) sum += blockShared[warp];


	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP PASS
	// Add sum to all the values in the continuing segment (that is, before the
	// first start flag) in this thread.

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		if(flags[i]) sum = 0;

		warpShared[offset + i] = sum;
		sum += x[i];
	}

	// Store the values back to global memory.
	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = threadShared[i * (WARP_SIZE + 1)];
		dataOut_global[index + i * WARP_SIZE] = x;
	}
}