// Demonstration of segmented scan. 
// See http://www.moderngpu.com/sparse/segscan.html

#define WARP_SIZE 32
#define LOG_WARP_SIZE 5

#define NUM_THREADS 256
#define NUM_WARPS 8
#define LOG_NUM_WARPS 3

#define VALUES_PER_THREAD 8
#define VALUES_PER_WARP (WARP_SIZE * VALUES_PER_THREAD)

#define DEVICE extern "C" __device__ __forceinline__


#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

typedef unsigned int uint;

DEVICE uint bfi(uint x, uint y, uint bit, uint numBits) {
	uint ret;
	asm("bfi.b32 %0, %1, %2, %3, %4;" : 
		"=r"(ret) : "r"(y), "r"(x), "r"(bit), "r"(numBits));
	return ret;
}


////////////////////////////////////////////////////////////////////////////////
// Use ballot and clz to run a segmented scan over a single warp, with one value
// per thread.

extern "C" __global__ void SegScanWarp(const uint* dataIn_global,
	uint* dataOut_global) {

	uint tid = threadIdx.x;
	uint packed = dataIn_global[tid];

	// The start flag is in the high bit.
	uint flag = 0x80000000 & packed;

	// Get the start flags for each thread in the warp.
	uint flags = __ballot(flag);

	// Mask out the bits above the current thread.
	flags &= bfi(0, 0xffffffff, 0, tid + 1);

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	uint distance =  __clz(flags) + tid - 31;

	__shared__ volatile uint shared[WARP_SIZE];

	uint x = 0x7fffffff & packed;
	uint x2 = x;
	shared[tid] = x;

	// Perform the parallel scan. Note the conditional if(offset < distance)
	// replaces the ordinary scan conditional if(offset <= tid).
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(offset <= distance) x += shared[tid - offset];
		shared[tid] = x;
	}

	// Turn inclusive scan into exclusive scan.
	x -= x2;

	dataOut_global[tid] = x;
}


////////////////////////////////////////////////////////////////////////////////
// Use parallel scan to compute the ranges for a segmented scan over a warp with
// eight values per thread.

extern "C" __global__ void SegScanWarp8(const uint* dataIn_global,
	uint* dataOut_global) {

	uint tid = threadIdx.x;
	
	__shared__ volatile uint shared[VALUES_PER_THREAD * (WARP_SIZE + 1)];
	
	// Load packed values from global memory and scatter to shared memory. Use
	// a 33-slot stride between successive values in each thread to set us up
	// for a conflict-free strided order -> thread order transpose.

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = dataIn_global[i * WARP_SIZE + tid];
		shared[i * (WARP_SIZE + 1) + tid] = x;
	}

	uint offset = VALUES_PER_THREAD * tid;
	offset += offset / WARP_SIZE;
	uint packed[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i)
		packed[i] = shared[offset + i];

	////////////////////////////////////////////////////////////////////////////
	// UPSWEEP PASS
	// Run a sequential segmented scan for all values in the packed array. Find
	// the sum of all values in the thread's last segment. Additionally set
	// index to tid if any segments begin in this thread.
	
	uint last = 0;
	uint hasHeadFlag = 0;

	uint scan[VALUES_PER_THREAD];
	uint flags[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		flags[i] = 0x80000000 & packed[i];
		uint x = 0x7fffffff & packed[i];
		if(flags[i]) last = 0;
		hasHeadFlag |= flags[i];
		scan[i] = last;
		last += x;
	}

	////////////////////////////////////////////////////////////////////////////
	// SEGMENT PASS
	// Run a ballot and clz to find the thread containing the start value for
	// the segment that begins this thread.

	uint warpFlags = __ballot(hasHeadFlag);

	// Mask out the bits at or above the current thread.
	warpFlags &= bfi(0, 0xffffffff, 0, tid);

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	int preceding = 31 - __clz(warpFlags);
	uint distance = tid - preceding;

	////////////////////////////////////////////////////////////////////////////
	// REDUCTION PASS
	// Run a prefix sum scan over last to compute for each thread the sum of all
	// values in the segmented preceding the current thread, up to that point.
	// This is added back into the thread-local exclusive scan for the continued
	// segment in each thread.

	shared[tid] = 0;
	volatile uint* shifted = shared + tid + 1;
	
	shifted[0] = last;
	uint sum = last;
	uint first = shared[1 + preceding];

	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(distance > offset) sum += shifted[-offset];
		shifted[0] = sum;
	}
	// Subtract last to make exclusive and add first to grab the fragment sum of
	// the preceding thread.
	sum += first - last;

	////////////////////////////////////////////////////////////////////////////
	// DOWNSWEEP PASS
	// Add sum to all the values in the continuing segment (that is, before the
	// first start flag) in this thread.

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		if(flags[i]) sum = 0;
		shared[offset + i] = scan[i] + sum;
	}

	// Store the values back to global memory.
	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = shared[i * (WARP_SIZE + 1) + tid];
		dataOut_global[i * WARP_SIZE + tid] = x;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Use a multiscan pattern to execute segmented scan over an entire block with
// 8 values per thread.

////////////////////////////////////////////////////////////////////////////////
// INTER-WARP REDUCTION 
// Calculate the length of the last segment in the last lane in each warp.

DEVICE uint BlockScan(uint warp, uint lane, uint last, uint warpFlags, 
	uint mask, volatile uint* shared, volatile uint* threadShared) {

	__shared__ volatile uint blockShared[3 * NUM_WARPS];
	if(WARP_SIZE - 1 == lane) {
		blockShared[NUM_WARPS + warp] = last;
		blockShared[2 * NUM_WARPS + warp] = warpFlags;
	}
	__syncthreads();

	if(lane < NUM_WARPS) {
		// Pull out the sum and flags for each warp.
		volatile uint* s = blockShared + NUM_WARPS + lane;
		uint warpLast = blockShared[NUM_WARPS + lane];
		uint flag = blockShared[2 * NUM_WARPS + lane];
		blockShared[lane] = 0;

		uint blockFlags = __ballot(flag);

		// Mask out the bits at or above the current warp.
		blockFlags &= mask;

		// Find the distance from the current warp to the warp at the start of 
		// this segment.
		int preceding = 31 - __clz(blockFlags);
		uint distance = lane - preceding;

		// INTER-WARP reduction
		uint warpSum = warpLast;
		uint warpFirst = blockShared[NUM_WARPS + preceding];

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			uint offset = 1<< i;
			if(distance > offset) warpSum += s[-offset];
			s[0] = warpSum;
		}
		// Subtract warpLast to make exclusive and add first to grab the
		// fragment sum of the preceding warp.
		warpSum += warpFirst - warpLast;

		// Store warpSum back into shared memory. This is added to all the
		// lane sums and those are added into all the threads in the first 
		// segment of each lane.
		blockShared[lane] = warpSum;
	}
	__syncthreads();

	return blockShared[warp];
}


extern "C" __global__ __launch_bounds__(NUM_THREADS, 4) 
void SegScanBlock8(const uint* dataIn_global, uint* dataOut_global) {

	uint tid = threadIdx.x;
	uint lane = (WARP_SIZE - 1) & tid;
	uint warp = tid / WARP_SIZE;
	
	const int Size = NUM_WARPS * VALUES_PER_THREAD * (WARP_SIZE + 1);
	__shared__ volatile uint shared[Size];

	// Use a stride of 33 slots per warp per value to allow conflict-free
	// transposes from strided to thread order.
	volatile uint* warpShared = shared + 
		warp * VALUES_PER_THREAD * (WARP_SIZE + 1);
	volatile uint* threadShared = warpShared + lane;
	

	////////////////////////////////////////////////////////////////////////////
	// Load packed values from global memory and scatter to shared memory. Use
	// a 33-slot stride between successive values in each thread to set us up
	// for a conflict-free strided order -> thread order transpose. Storing to
	// separate memory intervals allows use transpose without explicit
	// synchronization.

	uint index = VALUES_PER_WARP * warp + lane;

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = dataIn_global[index + i * WARP_SIZE];
		threadShared[i * (WARP_SIZE + 1)] = x;
	}

	uint offset = VALUES_PER_THREAD * lane;
	offset += offset / WARP_SIZE;
	uint packed[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i)
		packed[i] = warpShared[offset + i];


	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP UPSWEEP PASS
	// Run a sequential segmented scan for all values in the packed array. Find
	// the sum of all values in the thread's last segment. Additionally set
	// index to tid if any segments begin in this thread.
	
	uint last = 0;
	uint hasHeadFlag = 0;

	uint x[VALUES_PER_THREAD];
	uint flags[VALUES_PER_THREAD];

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		flags[i] = 0x80000000 & packed[i];
		x[i] = 0x7fffffff & packed[i];
		if(flags[i]) last = 0;
		hasHeadFlag |= flags[i];
		last += x[i];
	}


	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP SEGMENT PASS
	// Run a ballot and clz to find the lane containing the start value for
	// the segment that begins this thread.

	uint warpFlags = __ballot(hasHeadFlag);

	// Mask out the bits at or above the current thread.
	uint mask = bfi(0, 0xffffffff, 0, lane);
	uint warpFlagsMask = warpFlags & mask;

	// Find the distance from the current thread to the thread at the start of
	// the segment.
	int preceding = 31 - __clz(warpFlagsMask);
	uint distance = lane - preceding;


	////////////////////////////////////////////////////////////////////////////
	// REDUCTION PASS
	// Run a prefix sum scan over last to compute for each lane the sum of all
	// values in the segmented preceding the current lane, up to that point.
	// This is added back into the thread-local exclusive scan for the continued
	// segment in each thread.
	
	volatile uint* shifted = threadShared + 1;
	shifted[-1] = 0;
	shifted[0] = last;
	uint sum = last;
	uint first = warpShared[1 + preceding];

	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(distance > offset) sum += shifted[-offset];
		shifted[0] = sum;
	}
	// Subtract last to make exclusive and add first to grab the fragment sum of
	// the preceding thread.
	sum += first - last;


	// Call BlockScan for inter-warp scan on the reductions of the last segment
	// in each warp.
	uint lastSegLength = last;
	if(!hasHeadFlag) lastSegLength += sum;

	uint blockScan = BlockScan(warp, lane, lastSegLength, warpFlags, mask, 
		shared, threadShared);
	if(!warpFlagsMask) sum += blockScan;


	////////////////////////////////////////////////////////////////////////////
	// INTRA-WARP PASS
	// Add sum to all the values in the continuing segment (that is, before the
	// first start flag) in this thread.

	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		if(flags[i]) sum = 0;

		warpShared[offset + i] = sum;
		sum += x[i];
	}

	// Store the values back to global memory.
	#pragma unroll
	for(int i = 0; i < VALUES_PER_THREAD; ++i) {
		uint x = threadShared[i * (WARP_SIZE + 1)];
		dataOut_global[index + i * WARP_SIZE] = x;
	}
}