
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <vector_functions.h>

#define DEVICE extern "C" __forceinline__ __device__
typedef unsigned int uint;

#define ROUND_UP(x, y) (~(y - 1) & (x + y - 1))

#define WARP_SIZE 32
#define NUM_THREADS 256
#define BLOCKS_PER_SM 4
#define NUM_WARPS (NUM_THREADS / WARP_SIZE)

#define LOG_WARP_SIZE 5
#define LOG_NUM_THREADS 8
#define LOG_NUM_WARPS (LOG_NUM_THREADS - LOG_WARP_SIZE)

// Parameters for efficient sequential scan.
#define VALUES_PER_THREAD 8
#define NUM_VALUES (NUM_THREADS * VALUES_PER_THREAD)
#define SHARED_STRIDE (WARP_SIZE + 1)
#define SHARED_SIZE (NUM_VALUES + NUM_VALUES / WARP_SIZE)

__shared__ volatile uint values_shared[SHARED_SIZE];

////////////////////////////////////////////////////////////////////////////////
// Multiscan utility function. Used in the first and third passes of the
// global scan function. Returns the inclusive scan of the arguments in .x and
// the sum of all arguments in .y.

DEVICE uint2 Multiscan(uint tid, uint x) {
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	const int ScanStride = WARP_SIZE + WARP_SIZE / 2 + 1;
	const int ScanSize = NUM_WARPS * ScanStride;
	__shared__ volatile uint reduction_shared[ScanSize];
	__shared__ volatile uint totals_shared[NUM_WARPS + NUM_WARPS / 2];

	volatile uint* s = reduction_shared + ScanStride * warp + lane + 
		WARP_SIZE / 2;
	s[-16] = 0;
	s[0] = x;

	// Run inclusive scan on each warp's data.
	uint sum = x;	
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		sum += s[-offset];
		s[0] = sum;
	}

	// Synchronize to make all the totals available to the reduction code.
	__syncthreads();
	if(tid < NUM_WARPS) {
		// Grab the block total for the tid'th block. This is the last element
		// in the block's scanned sequence. This operation avoids bank 
		// conflicts.
		uint total = reduction_shared[ScanStride * tid + WARP_SIZE / 2 +
			WARP_SIZE - 1];

		totals_shared[tid] = 0;
		volatile uint* s2 = totals_shared + NUM_WARPS / 2 + tid;
		uint totalsSum = total;
		s2[0] = total;

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			int offset = 1<< i;
			totalsSum += s2[-offset];
			s2[0] = totalsSum;	
		}

		// Subtract total from totalsSum for an exclusive scan.
		totals_shared[tid] = totalsSum - total;
	}

	// Synchronize to make the block scan available to all warps.
	__syncthreads();

	// Add the block scan to the inclusive sum for the block.
	sum += totals_shared[warp];
	uint total = totals_shared[NUM_WARPS + NUM_WARPS / 2 - 1];
	return make_uint2(sum, total);
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanPass1 adds up all the values in elements_global within the 
// range given by blockCount and writes to blockTotals_global[blockIdx.x].

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanPass1(const uint* elements_global, const uint2* range_global,
	uint* blockTotals_global) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	uint2 range = range_global[block];

	// Loop through all elements in the interval, adding up values.
	// There is no need to synchronize until we perform the multiscan.
	uint sum = 0;
	for(uint index = range.x + tid; index < range.y; index += NUM_THREADS)
		sum += elements_global[index];
	
	// A full multiscan is unnecessary here - we really only need the total.
	// But this is easy and won't slow us down since this kernel is already
	// bandwidth limited.
	uint total = Multiscan(tid, sum).y;

	// The last scan element in the block is the total for all values summed
	// in this block.
	if(tid == NUM_THREADS - 1)
		blockTotals_global[block] = total;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanPass2 performs an exclusive scan on the elements in 
// blockTotals_global and writes back in-place.

extern "C" __global__ void GlobalScanPass2(uint* blockTotals_global, 
	uint numBlocks) {

	uint tid = threadIdx.x;
	uint x = 0; 
	if(tid < numBlocks) x = blockTotals_global[tid];

	// Subtract the value from the inclusive scan for the exclusive scan.
	uint2 scan = Multiscan(tid, x);
	if(tid < numBlocks) blockTotals_global[tid] = scan.x - x;
	
	// Have the first thread in the block set the scan total.
	if(!tid) blockTotals_global[numBlocks] = scan.y;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanPass3 runs an exclusive scan on the same interval of data as in
// pass 1, and adds blockScan_global[blockIdx.x] to each of them, writing back
// out in-place.

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanPass3(uint* elements_global, const uint2* range_global,
	uint* blockScan_global, int inclusive) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	uint blockScan = blockScan_global[block];
	uint2 range = range_global[block];

	// Have each warp read a consecutive block of memory. Because threads in a
	// warp are implicitly synchronized, we can "transpose" the terms into
	// thread-order without a __syncthreads().
	uint first = range.x + warp * (VALUES_PER_THREAD * WARP_SIZE) + lane;
	uint end = ROUND_UP(range.y, NUM_VALUES);

	// Get a pointer to the start of this warp's shared memory storage for 
	// value transpose.
	volatile uint* warpValues = values_shared +
		warp * SHARED_STRIDE * VALUES_PER_THREAD;

	// The threads write to threadValues[i * SHARED_STRIDE]
	volatile uint* threadValues = warpValues + lane;

	// The threads read from transposeValues[i]
	uint valueOffset = lane * VALUES_PER_THREAD;
	volatile uint* transposeValues = warpValues + valueOffset + 
		valueOffset / WARP_SIZE;
	
	for(uint index = first; index < end; index += NUM_VALUES) {

		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint index2 = index + i * WARP_SIZE;
			uint value = 0;
			if(index2 < range.y) value = elements_global[index2];
		
			threadValues[i * SHARED_STRIDE] = value;
		}

		// Transpose into thread order by reading from transposeValues.
		// Compute the exclusive or inclusive scan of the thread values and 
		// their sum.
		uint scan[VALUES_PER_THREAD];
		uint sum = 0;
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = transposeValues[i];
			scan[i] = sum;
			if(inclusive) scan[i] += x;
			sum += x;
		}

		// Multiscan for each thread's scan offset within the block. Subtract
		// sum to make it an exclusive scan.
		uint2 localScan = Multiscan(tid, sum);
		uint scanOffset = localScan.x + blockScan - sum;

		// Add the scan offset to each exclusive scan and put the values back
		// into the shared memory they came out of.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = scan[i] + scanOffset;
			transposeValues[i] = x;
		}

		// Store the scan back to global memory.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = threadValues[i * SHARED_STRIDE];
			uint index2 = index + i * WARP_SIZE;			
			if(index2 < range.y) elements_global[index2] = x;
		}

		// Grab the last element of totals_shared, which was set in Multiscan.
		// This is the total for all the values encountered in this pass.
		blockScan += localScan.y;
	}
}

