// Demonstration of ballot scan. 
// See http://www.moderngpu.com/intro/scan.html#Ballot

#define WARP_SIZE 32
#define LOG_WARP_SIZE 5

#define NUM_THREADS 256
#define NUM_WARPS 8
#define LOG_NUM_WARPS 3

#define DEVICE extern "C" __device__ __forceinline__


#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

typedef unsigned int uint;

DEVICE uint bfi(uint x, uint y, uint bit, uint numBits) {
	uint ret;
	asm("bfi.b32 %0, %1, %2, %3, %4;" : 
		"=r"(ret) : "r"(y), "r"(x), "r"(bit), "r"(numBits));
	return ret;
}



////////////////////////////////////////////////////////////////////////////////
// Use parallel scan for stream compaction. All values that are not -1.0f are
// moved to the front of the stream and stored to dataOut_global. The total
// number of defined values is stored in countOut_global.

extern "C" __global__ void ParallelScanWarp(const float* dataIn_global, 
	float* dataOut_global, uint* countOut_global) {

	__shared__ volatile uint shared[WARP_SIZE + WARP_SIZE / 2];

	uint tid = threadIdx.x;
	float val = dataIn_global[tid];

	shared[tid] = 0;

	volatile uint* s = shared + tid + WARP_SIZE / 2;

	// Scan the number of non -1.0 elements.
	uint flag = -1.0f != val;
	uint x = flag;
	s[0] = x;

	// Run a parallel scan.
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		uint y = s[-offset];
		x += y;
		s[0] = x;
	}

	// Subtract the flag to get an exclusive scan.
	uint scan = x - flag;
	uint total = shared[WARP_SIZE + WARP_SIZE / 2 - 1];

	volatile float* s2 = (volatile float*)shared;
	if(flag) s2[scan] = val;

	val = s2[tid];

	if(tid < total)
		dataOut_global[tid] = val;

	if(!tid) 
		*countOut_global = total;
}


////////////////////////////////////////////////////////////////////////////////
// Use ballot scan (ballot-mask-popc) for a fast parallel scan on one-bit
// sequences.

extern "C" __global__ void BallotScanWarp(const float* dataIn_global, 
	float* dataOut_global, uint* countOut_global) {

	uint tid = threadIdx.x;
	float val = dataIn_global[tid];

	uint flag = -1.0f != val;

	uint bits = __ballot(flag);

	uint mask = bfi(0, 0xffffffff, 0, tid);
	uint exc = __popc(mask & bits);
	uint total = __popc(bits);

	__shared__ volatile float shared[WARP_SIZE];
	if(flag) shared[exc] = val;
	val = shared[tid];

	if(tid < total) 
		dataOut_global[tid] = val;

	if(!tid) *countOut_global = total;
}



////////////////////////////////////////////////////////////////////////////////
// Multiscan utility function. Returns the inclusive scan of the arguments in 
// .x and the sum of all arguments in .y.

DEVICE uint2 Multiscan(uint tid, uint x) {
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	const int ScanStride = WARP_SIZE + WARP_SIZE / 2 + 1;
	const int ScanSize = NUM_WARPS * ScanStride;
	__shared__ volatile uint reduction_shared[ScanSize];
	__shared__ volatile uint totals_shared[NUM_WARPS + NUM_WARPS / 2];

	volatile uint* s = reduction_shared + ScanStride * warp + lane + 
		WARP_SIZE / 2;
	s[-16] = 0;
	s[0] = x;

	// Run inclusive scan on each warp's data.
	uint sum = x;	
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		sum += s[-offset];
		s[0] = sum;
	}

	// Synchronize to make all the totals available to the reduction code.
	__syncthreads();
	if(tid < NUM_WARPS) {
		// Grab the block total for the tid'th block. This is the last element
		// in the block's scanned sequence. This operation avoids bank 
		// conflicts.
		uint total = reduction_shared[ScanStride * tid + WARP_SIZE / 2 +
			WARP_SIZE - 1];

		totals_shared[tid] = 0;
		volatile uint* s2 = totals_shared + NUM_WARPS / 2 + tid;
		uint totalsSum = total;
		s2[0] = total;

		#pragma unroll
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			int offset = 1<< i;
			totalsSum += s2[-offset];
			s2[0] = totalsSum;	
		}

		// Subtract total from totalsSum for an exclusive scan.
		totals_shared[tid] = totalsSum - total;
	}

	// Synchronize to make the block scan available to all warps.
	__syncthreads();

	// Add the block scan to the inclusive sum for the block.
	sum += totals_shared[warp];
	uint total = totals_shared[NUM_WARPS + NUM_WARPS / 2 - 1];
	return make_uint2(sum, total);
}


////////////////////////////////////////////////////////////////////////////////

extern "C" __global__ void ParallelScanBlock(const float* dataIn_global, 
	float* dataOut_global, uint* countOut_global) {

	uint tid = threadIdx.x;
	float val = dataIn_global[tid];

	uint flag = -1.0f != val;

	uint2 scan = Multiscan(tid, flag);
	
	__shared__ volatile float shared[NUM_THREADS];
	uint exc = scan.x - flag;

	if(flag) shared[exc] = val;
	__syncthreads();
	
	if(tid < scan.y) {
		val = shared[tid];
		dataOut_global[tid] = val;
	}

	if(!tid) 
		*countOut_global = scan.y;
}

extern "C" __global__ void BallotScanBlock(const float* dataIn_global, 
	float* dataOut_global, uint* countOut_global) {

	uint tid = threadIdx.x;
	uint lane = (WARP_SIZE - 1) & tid;
	uint warp = tid / WARP_SIZE;

	float val = dataIn_global[tid];

	uint flag = -1.0f != val;

	uint bits = __ballot(flag);

	uint mask = bfi(0, 0xffffffff, 0, lane);
	uint exc = __popc(mask & bits);
	uint warpTotal = __popc(bits);

	__shared__ volatile uint shared[NUM_WARPS];
	if(!lane) shared[warp] = warpTotal;

	// Inclusive scan the warp totals.
	__syncthreads();
	if(tid < NUM_WARPS) {
		uint x = shared[tid];
		for(int i = 0; i < LOG_NUM_WARPS; ++i) {
			uint offset = 1<< i;
			if(tid >= offset) x += shared[tid - offset];
			shared[tid] = x;
		}
	}
	__syncthreads();

	// Add the scanned warp totals into exc.
	uint blockTotal = shared[NUM_WARPS - 1];
	exc += shared[warp] - warpTotal;

	__shared__ volatile float shared2[NUM_THREADS];
	if(flag) shared2[exc] = val;
	
	__syncthreads();

	dataOut_global[tid] = exc;

	if(tid < blockTotal) {
		val = shared2[tid];
		dataOut_global[tid] = val;
	}

	if(!tid) 
		*countOut_global = blockTotal;
}
