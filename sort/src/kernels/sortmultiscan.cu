#include "hip/hip_runtime.h"
#pragma once

#include "common.cu"

// counterSize is 8 (for byte-packed) or 16 (for short-packed).
// strided is false to put digits in order (0, 1), (2, 3), (4, 5), etc.
// strided is true to put digits in order (0, 16), (1, 17), (2, 18), etc.


template<int NumDigits, int NumThreads>
DEVICE2 volatile uint* PackedCounterRef(uint digit, uint tid, 
	volatile uint* counters_shared, int counterSize, bool strided, 
	uint& shift) {

	const int NumBits = LogPow2Const<NumDigits>::value;
	const int LogThreads = LogPow2Const<NumThreads>::value;

	volatile uint* counter;
	if(8 == counterSize) {
		if(strided) {
			uint index = NumThreads * ((DIV_UP(NumDigits, 4) - 1) & digit);
			shift = 8 * (digit / DIV_UP(NumDigits,  4));
			counter = &counters_shared[index + tid];
		} else {
			uint index = NumThreads * (digit / 4);
			shift = 8 * (3 & digit);
			counter = &counters_shared[index + tid];
		}
	} else if(16 == counterSize) { 
		if(strided) {
			// If NumDigits is 32 we can just mask out the least sig four bits
			// to accelerate 16 * (digit / (NumDigits / 2)).
			if(32 == NumDigits) shift = 16 & digit;
			else shift = 16 * (digit / (NumDigits / 2));
		
			// uint index = 16 * (digit / (NumDigits / 2));
			// We can accelerate that by using bfi to insert the least sig bits
			// of the digit into the pre-scaled tid. Note that we need to pass
			// counters_shared as a CONSTANT pointer from the start of the shmem
			// array for the block, as Fermi doesn't allow you to add two 
			// registers together in an LDS or STS statement.
			uint index = bfi(4 * tid, digit, 2 + LogThreads, NumBits - 1);
			counter = (volatile uint*)((volatile char*)counters_shared + index);
		} else {
			uint index = NumThreads * (digit / 2);
			shift = 16 * (1 & digit);
			counter = &counters_shared[index + tid];
		} 
	}
	return counter;
}

// Returns the counter content before the increment.
template<int NumDigits, int NumThreads>
DEVICE2 uint IncPackedCounter(uint digit, uint tid,
	volatile uint* counters_shared, int counterSize, bool strided, 
	uint value, uint& shift) {

	volatile uint* p = PackedCounterRef<NumDigits, NumThreads>(digit, tid,
		counters_shared, counterSize, strided, shift);
	uint counter = *p;
	*p = shl_add(value, shift, counter);
	return counter;
}

template<int NumDigits, int NumThreads>
DEVICE2 uint GatherPackedCounter(uint digit, uint tid,
	volatile uint* counters_shared, int counterSize, bool strided, 
	uint& shift) {

	volatile uint* p = PackedCounterRef<NumDigits, NumThreads>(digit, tid,
		counters_shared, counterSize, strided, shift);
	return *p;
}


////////////////////////////////////////////////////////////////////////////////
// MultiscanParams

// Parameters and sizes for multiscan.

template<int NumThreads, int NumBits>
struct MultiscanParams {

	static const int NumDigits = 1<< NumBits;
	static const int NumCounters = NumDigits / 2;

	static const int TotalCounters = NumThreads * NumCounters;
	static const int SegLen = TotalCounters / NumThreads + 1;

	// Each raking thread processes SegLen counters, so NumScanValues is total
	// number of raking scan values. These values are colocated with the 
	// column counters.
	static const int NumScanValues = NumThreads * SegLen;
	static const int NumRedValues = NumThreads / WARP_SIZE;

	static const int TotalRedValues = WARP_SIZE * NumRedValues;
	static const int RedFootprint = TotalRedValues + TotalRedValues / WARP_SIZE;

	static const int ParallelScanSize = WARP_SIZE + WARP_SIZE / 2;

	// Allocate at least this much shared memory per block to facilitate the
	// parallel multiscan.
	static const int ScratchSize = NumScanValues + RedFootprint +
		ParallelScanSize;
};


////////////////////////////////////////////////////////////////////////////////


// recalcCounts = false: keep the thread-local per-digit scans in register.
// recalcCounts = true: discard the thread-local per-digit scans prior to
//		the parallel scan. The recompute the histogram counts after the
//		parallel scan. This reduces register pressure.

// reloadParallel = false: keep the multiple elements loaded per thread in the
//		parallel scan in register.
// reloadParallel = true: reload the multiple elements per thread in the 
//		parallel scan to reduce register pressure.
template<int NumThreads, int NumBits, int ValuesPerThread>
DEVICE2 void MultiScanCounters(uint tid, const uint* keys, uint bit, 
	volatile uint* scratch_shared, uint* scatter, bool recalcCounts,
	bool recalcDigits, bool reloadParallel) {

	// Evenly divide the TotalCounters counters over all raking threads..
	// However add 1 to make SegLen relatively prime to the number of banks to
	// avoid bank conflicts. For NumDigits = 32, NumCounters = 16, and 
	// NumThreads = NumRakingThreads = 128, each thread sequentially scans 16
	// counters, but this number is bumped up by 1:

	// eg,
	// tid 0 processes bank 0 on cycle 0
	// tid 1 processes bank 17 on cycle 0
	// tid 2 processes bank 2 on cycle 0
	// tid 3 process bank 19 on cycle 0, etc.

	// Parallel scan is handled from a single warp. 128 reduction totals are 
	// stored to shared memory, and strided. Each thread of warp 0 loads four
	// of these values, sums them up, and runs a simple 5-level parallel scan.
	// The four reduction values are then subtracted out in reverse order, and
	// each difference is stored back to the shared mem location from which its
	// reduction value was loaded.

	// For a block with 128 threads and 16 values per thread, these 2048 values
	// generate 2048 digit counters (for NumBits = 5), but they are scanned with
	// sequential scans and just a single simple intra-warp parallel scan.

	typedef MultiscanParams<NumThreads, NumBits> Params;
	const int NumDigits = 1<< NumBits;
	const int NumCounters = NumDigits / 2;
	const int NumScanValues = Params::NumScanValues;
	const int NumRedValues = Params::NumRedValues;
	const int SegLen = Params::SegLen;

	uint warp = tid / WARP_SIZE;

	volatile uint* counters_shared = scratch_shared;
	volatile uint* reduction_shared = scratch_shared + NumScanValues;
	volatile uint* scan_shared = reduction_shared + Params::RedFootprint;

	// Clear the counters.
	#pragma unroll
	for(int i = 0; i < NumCounters; ++i)
		counters_shared[i * NumThreads + tid] = 0;

	// Clear the padding counters at the end.
	scratch_shared[SegLen * NumThreads - NumThreads + tid] = 0;


	// Compute the digit counts and save the thread-local scan per digit.
	// NOTE: may need to bfi into localScan to fight register pressure.
	// Shift and add in a 4 for each digit occurence to eliminate some pointer
	// arithmetic due to Fermi not having a mov/lea-like 4 * mul in STS/LDS.
	uint digits[ValuesPerThread];
	uint localScan[ValuesPerThread];
	#pragma unroll
	for(int v = 0; v < ValuesPerThread; ++v) {
		digits[v] = bfe(keys[v], bit, NumBits);
		uint shift;
		localScan[v] = IncPackedCounter<NumDigits, NumThreads>(digits[v], tid,
			counters_shared, 16, true, 4, shift);
	}
	__syncthreads();

	// Add up all the packed counters in this segment. We would prefer to load
	// them once and create an exclusive sequential scan in register array, but
	// there aren't enough registers to allow this. Instead, we load them a 
	// second time after the parallel scan and do the running sum.
	volatile uint* seg_shared = scratch_shared + SegLen * tid;
/*	uint x = 0;
	#pragma unroll
	for(int i = 0; i < SegLen; ++i)
		x += seg_shared[i];
	*/
	uint x = seg_shared[0];
	if(2 == SegLen) x += seg_shared[1];
	else {
		#pragma unroll
		for(int i = 1; i < SegLen; i += 2) {
			uint a = seg_shared[i];
			uint b = seg_shared[i + 1];
			x = add3(x, a, b);
		}
	}

	// Store the counters with stride.
	reduction_shared[tid + warp] = x;

	__syncthreads();

	// Scan from a single warp.
	if(tid < WARP_SIZE) {
		uint index = NumRedValues * tid;
		index += index / WARP_SIZE;
		
		uint threadVals[NumRedValues];
		uint sum = 0;
		#pragma unroll
		for(int i = 0; i < NumRedValues; ++i) {
			threadVals[i] = reduction_shared[index + i];
			sum += threadVals[i];
		}

		// Run a single parallel scan.
		volatile uint* s = scan_shared + tid + WARP_SIZE / 2;
		s[-(WARP_SIZE / 2)] = 0;
		s[0] = sum;

		uint x = sum;
		#pragma unroll
		for(int i = 0; i < LOG_WARP_SIZE; ++i) {
			int offset = 1<< i;
			x += s[-offset];
			s[0] = x;
		}

		// Add in the reduction of the top row to all elements in the bottom
		// row of the packed scan.
		x += scan_shared[WARP_SIZE + WARP_SIZE / 2 - 1]<< 16;

		// Subtract out the threadVals to get an exclusive scan and store.
		if(reloadParallel) {
			#pragma unroll
			for(int i = NumRedValues - 1; i >= 0; --i) {
				x -= reduction_shared[index + i];
				reduction_shared[index + i] = x;
			}
		} else {
			#pragma unroll
			for(int i = NumRedValues - 1; i >= 0; --i) {
				x -= threadVals[i];
				reduction_shared[index + i] = x;
			}
		}
	}
	__syncthreads();

	// Add the scanned values back into the stored scanValues.
	x = 0;
	uint offset = reduction_shared[tid + warp];
	#pragma unroll
	for(int i = 0; i < SegLen; ++i)  {
		uint scanValue = seg_shared[i];
		seg_shared[i] = x + offset;
		x += scanValue;
	}
	__syncthreads();

	// Gather the scanned offsets for each digit and add in the local offsets
	// saved in localScan. We'd rather make this switch inside the loop, but
	// the open64 compiler gives "Advisory: Loop was not unrolled, unexpected 
	// control flow construct" warnings.
	if(recalcCounts) {
		#pragma unroll
		for(int v = 0; v < ValuesPerThread; ++v) {
			if(recalcDigits) digits[v] = bfe(keys[v], bit, NumBits);
				
			uint shift;
			uint offset = IncPackedCounter<NumDigits, NumThreads>(digits[v], 
				tid, counters_shared, 16, true, 4, shift);

			scatter[v] = bfe(offset, shift, 16);
		}
	} else {
		#pragma unroll
		for(int v = 0; v < ValuesPerThread; ++v) {
			if(recalcDigits) digits[v] = bfe(keys[v], bit, NumBits);
				
			uint shift;
			uint offset = GatherPackedCounter<NumDigits, NumThreads>(
				digits[v], tid, counters_shared, 16, true, shift);

			scatter[v] = bfe(localScan[v] + offset, shift, 16);
		}
	}

	__syncthreads();
}


////////////////////////////////////////////////////////////////////////////////
//


template<int NumThreads, int NumBits, int ValuesPerThread>
DEVICE2 void LocalSort(uint tid, uint* keys, uint bit, 
	volatile uint* scratch_shared, uint* scatter, bool recalcCounts,
	bool recalcDigits, bool reloadParallel) {

}