#include "hip/hip_runtime.h"
#pragma once

#include "sortcommon.cu"

////////////////////////////////////////////////////////////////////////////////
// SortLocal

// Given keys in thread order (fusedKeys) or keys in shared memory in strided
// order (scattergather_shared), sort between 1 and 7 key bits and store into
// shared memory.

template<int NumThreads, int NumBits>
DEVICE2 void SortLocal(uint tid, Values fusedKeys, uint bit, bool loadFromArray,
	uint* scattergather_shared, uint* scratch_shared, uint* debug_global) {

	if(1 == NumBits) 
		SortAndScatter(tid, fusedKeys, bit, 1, NumThreads, loadFromArray,
			false, scattergather_shared, scratch_shared, debug_global);
	else if(2 == NumBits)
		SortAndScatter(tid, fusedKeys, bit, 2, NumThreads, loadFromArray,
			false, scattergather_shared, scratch_shared, debug_global);
	else if(3 == NumBits)
		SortAndScatter(tid, fusedKeys, bit, 3, NumThreads, loadFromArray,
			false, scattergather_shared, scratch_shared, debug_global);
	else if(4 == NumBits) {
		/*SortAndScatter(tid, fusedKeys, bit, 4, NumThreads,
			!LoadFromTexture, false, scattergather_shared, scratch_shared, 
			debug_global_out);*/
		SortAndScatter(tid, fusedKeys, bit, 2, NumThreads, loadFromArray, 
			true, scattergather_shared, scratch_shared, debug_global);
		SortAndScatter(tid, fusedKeys, bit + 2, 2, NumThreads, true, false,
			scattergather_shared, scratch_shared, debug_global);
	} else if(5 == NumBits) {
		SortAndScatter(tid, fusedKeys, bit, 2, NumThreads, loadFromArray, 
			true, scattergather_shared, scratch_shared, debug_global);
		SortAndScatter(tid, fusedKeys, bit + 2, 3, NumThreads, true, false,
			scattergather_shared, scratch_shared, debug_global);
	} else if(6 == NumBits) {
		SortAndScatter(tid, fusedKeys, bit, 3, NumThreads, loadFromArray, 
			true, scattergather_shared, scratch_shared, debug_global);
		SortAndScatter(tid, fusedKeys, bit + 3, 3, NumThreads, true, false,
			scattergather_shared, scratch_shared, debug_global);
	} else if(7 == NumBits) {
		SortAndScatter(tid, fusedKeys, bit, 2, NumThreads, loadFromArray, 
			true, scattergather_shared, scratch_shared, debug_global);
		SortAndScatter(tid, fusedKeys, bit + 2, 2, NumThreads, true, true,
			scattergather_shared, scratch_shared, debug_global);
		SortAndScatter(tid, fusedKeys, bit + 4, 3, NumThreads, true, false,
			scattergather_shared, scratch_shared, debug_global);
	}
}


////////////////////////////////////////////////////////////////////////////////

template<int NumThreads, int NumBits, int ValueCount, bool UseScatterList,
	bool LoadFromTexture, bool DetectEarlyExit>
DEVICE2 void SortFunc(const uint* keys_global_in, uint firstBlock,
	const uint* bucketCodes_global, uint bit, uint* keys_global_out,
	uint numValueStreams, uint* debug_global_out, 
	const uint* values1_global_in, const uint* values2_global_in,
	const uint* values3_global_in, const uint* values4_global_in,
	const uint* values5_global_in, const uint* values6_global_in,
	// For VALUE_TYPE_MULTI, we have to pass each of the pointers in as 
	// individual arguments, not as arrays, or CUDA generates much worse code,
	// full of unified addressing instructions.
	uint* values1_global_out, uint* values2_global_out,
	uint* values3_global_out, uint* values4_global_out,
	uint* values5_global_out, uint* values6_global_out) {

	const int NumValues = VALUES_PER_THREAD * NumThreads;
	const int NumWarps = NumThreads / WARP_SIZE;
	const int Stride = WARP_SIZE + 1;

	const int NumBuckets = 1<< NumBits;

	// Simple scatter
	const int ScratchSize = 2 * (NumThreads + NumWarps) + 4 * WARP_SIZE + 32;

	__shared__ uint scratch_shared[ScratchSize];

	const int ScatterStructSize = NumBuckets;
	
	__shared__ uint scatterList_shared[ScatterStructSize];
	__shared__ uint scattergather_shared[NumWarps * VALUES_PER_THREAD * Stride];


	////////////////////////////////////////////////////////////////////////////
	// LOAD KEYS, CREATE FUSED KEYS, AND REINDEX INTO THREAD ORDER

	uint tid = threadIdx.x;
	uint block = blockIdx.x + firstBlock;
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	debug_global_out += NumValues * block;

	// Load the scatter (transaction) structure.
	uint globalStructOffset = ScatterStructSize * block;
	if(tid < ScatterStructSize)
		scatterList_shared[tid] = 
			bucketCodes_global[globalStructOffset + tid];

	// Load the keys and, if sorting values, create fused keys. Store into 
	// shared mem with a WARP_SIZE + 1 stride between warp rows, so that loads
	// into thread order occur without bank conflicts.
	Values keys, fusedKeys;

	if(LoadFromTexture) {
		// Load keys from a texture. The texture sampler serves as an 
		// asynchronous independent subsystem. It helps transpose data from
		// strided to thread order without involving the shader units.

		uint keysOffset = NumValues * blockIdx.x + VALUES_PER_THREAD * tid;

		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD / 4; ++i) {
			uint4 k = tex1Dfetch(keys_texture_in, keysOffset / 4 + i);
			keys[4 * i + 0] = k.x;
			keys[4 * i + 1] = k.y;
			keys[4 * i + 2] = k.z;
			keys[4 * i + 3] = k.w;
		}

		if(0 == ValueCount)
			// Sort only keys.
			#pragma unroll
			for(int i = 0; i < VALUES_PER_THREAD; ++i)
				fusedKeys[i] = keys[i];
		else
			// Sort key-value tuples.
			BuildFusedKeysThreadOrder(tid, keys, bit, NumBits, fusedKeys,
				false);

	} else {
		// Load keys from global memory. This requires using shared memory to 
		// transpose data from strided to thread order.

		LoadWarpValues(keys_global_in, warp, lane, block, keys);

		if(0 == ValueCount)
			// Sort only keys.
			#pragma unroll
			for(int i = 0; i < VALUES_PER_THREAD; ++i)
				fusedKeys[i] = keys[i];
		else
			// Sort key-value tuples.
			BuildFusedKeysWarpOrder(warp, lane, keys, bit, NumBits, fusedKeys,
				false);

		// Store the keys or fused keys into shared memory for the
		// strided->thread order transpose.
		ScatterWarpOrder(warp, lane, true, fusedKeys, scattergather_shared);
	}


	////////////////////////////////////////////////////////////////////////////
	// Check the early exit code for this block.

	bool isEarlyDetect = false;
/*	if(DetectEarlyExit) {
		if(!tid) {
			uint scatter = scatterList_shared[0];
			scratch_shared[0] = 1 & scatter;
			scatterList_shared[0] = ~1 & scatter;		
		}
		__syncthreads();
		isEarlyDetect = scratch_shared[0];
	}*/

	// Sort the fused keys in shared memory if early exit was not detected.
	if(!isEarlyDetect) {
		uint scanBitOffset = ValueCount ? 24 : bit;

		SortLocal<NumThreads, NumBits>(tid, fusedKeys, scanBitOffset,
			!LoadFromTexture, scattergather_shared, scratch_shared, 
			debug_global_out);
	} else if(LoadFromTexture) {
		// Copy the data fusedKeys to shared memory?


	}


	////////////////////////////////////////////////////////////////////////////
	// Store the keys and values to global memory.

	if(0 == ValueCount) {


	//	#pragma unroll
	//	for(int v = 0; v < VALUES_PER_THREAD; ++v)
	//		keys_global_out[NumValues * block + NumThreads * v + tid] =
	//			scattergather_shared[NumThreads * v + tid];

		// Store only keys.
		GatherBlockOrder(tid, false, NumThreads, keys, scattergather_shared);
		ScatterKeysSimple(tid, keys_global_out, bit, NumBits, 
			scatterList_shared, keys);

	} else if(-1 == ValueCount) {
		// Store keys and indices.
/*
		// Read the fused keys from shared memory into register and break into
		// pre-multiplied gather indices.
		Values gather;
		GatherBlockOrder(tid, true, fusedKeys);
		BuildGatherFromFusedKeys(fusedKeys, gather);
		__syncthreads();

		// Store the keys to shared memory without padding.
		ScatterWarpOrder(warp, lane, false, keys);
		__syncthreads();

		// Gather the keys from shared memory.
		GatherFromIndex(gather, true, keys);

		// Generate index values from the gather indices. As these are 
		// pre-multiplied, divide them by 4 and add the block offset.
		uint blockOffset = NUM_VALUES * block;
		Values indices;
		#pragma unroll
		for(int v = 0; v < VALUES_PER_THREAD; ++v)
			indices[v] = shr_add(gather[v], 2, blockOffset);

		ScatterPairSimple(tid, keys_global_out, index_global_out, bit, NUM_BITS,
			(const uint*)compressedList_shared, keys, indices);*/

	} else if(1 == ValueCount) {
		// Store key-value pairs.

		/*
		Values values;
		LoadBlockValues(value1_global_in, tid, block, values);
	
		// Read the fused keys from shared memory into register and break into
		// pre-multiplied gather indices.
		Values gather;
		GatherBlockOrder(tid, true, fusedKeys);
		BuildGatherFromFusedKeys(fusedKeys, gather);
		__syncthreads();

		// Store the keys to shared memory without padding.
		ScatterWarpOrder(warp, lane, false, keys);
		__syncthreads();

		// Gather the keys from shared memory.
		GatherFromIndex(gather, true, keys);
		__syncthreads();

		// Store the values to shared memory.
		ScatterBlockOrder(tid, false, values);
		// ScatterWarpOrder(warp, lane, false, values);
		__syncthreads();

		// Gather the values from shared memory.
		GatherFromIndex(gather, true, values);

		ScatterPairSimple(tid, keys_global_out, value1_global_out, bit, 
			NUM_BITS, (const uint*)compressedList_shared, keys, values);
*/
	} else {
		// Store keys with multiple value streams.

		// Read the fused keys from shared memory into register and break into
		// pre-multiplied gather indices.
		/*
		Values gather;
		GatherBlockOrder(tid, true, fusedKeys);
		BuildGatherFromFusedKeys(fusedKeys, gather);
		__syncthreads();

		// Store the keys to shared memory without padding.
		ScatterWarpOrder(warp, lane, false, keys);
		__syncthreads();

		// Gather the keys from shared memory.
		GatherFromIndex(gather, true, keys);
		__syncthreads();

		Values globalScatter;
		MultiScatterSimple(tid, keys_global_out, bit, NUM_BITS,
			(const uint*)compressedList_shared, keys, globalScatter);

		GlobalGatherScatter(tid, block, values1_global_in, 
			values1_global_out, gather, globalScatter);

		GlobalGatherScatter(tid, block, values2_global_in, 
			values2_global_out, gather, globalScatter);

		if(numValueStreams >= 3)
			GlobalGatherScatter(tid, block, values3_global_in, 
				values3_global_out, gather, globalScatter);

		if(numValueStreams >= 4)
			GlobalGatherScatter(tid, block, values4_global_in, 
				values4_global_out, gather, globalScatter);

		if(numValueStreams >= 5)
			GlobalGatherScatter(tid, block, values5_global_in, 
				values5_global_out, gather, globalScatter);

		if(6 == numValueStreams)
			GlobalGatherScatter(tid, block, values6_global_in, 
				values6_global_out, gather, globalScatter);
*/
	}


}

/*
#define GEN_SORT_FUNC(Name, NumThreads, NumBits, ValueCount,				\
	UseScatterList, LoadFromTexture, EarlyExit, BlocksPerSM)				\
																			\
extern "C" __global__ __launch_bounds__(NumThreads, BlocksPerSM)			\
void Name(const uint* keys_global_in, uint firstBlock,						\
	const uint* bucketCodes_global, uint bit, uint* keys_global_out,		\
	uint numValueStreams, uint* debug_global_out,							\
	const uint* values1_global_in, const uint* values2_global_in,			\
	const uint* values3_global_in, const uint* values4_global_in,			\
	const uint* values5_global_in, const uint* values6_global_in,			\
	uint* values1_global_out, uint* values2_global_out,						\
	uint* values3_global_out, uint* values4_global_out,						\
	uint* values5_global_out, uint* values6_global_out) {					\
																			\
	SortFunc<NumThreads, NumBits, ValueCount, UseScatterList,				\
		LoadFromTexture, EarlyExit>(										\
		keys_global_in, firstBlock, bucketCodes_global, bit,				\
		keys_global_out, numValueStreams, debug_global_out,					\
		values1_global_in, values2_global_in, values3_global_in,			\
		values4_global_in, values5_global_in, values6_global_in,			\
		values1_global_out, values2_global_out, values3_global_out,			\
		values4_global_out, values5_global_out, values6_global_out);		\
}*/

#define GEN_SORT_FUNC(Name, NumThreads, NumBits, ValueCount,				\
	UseScatterList, LoadFromTexture, EarlyExit, BlocksPerSM)				\
																			\
extern "C" __global__ __launch_bounds__(NumThreads, BlocksPerSM)			\
void Name(const uint* keys_global_in, uint firstBlock,						\
	const uint* bucketCodes_global, uint bit, uint* keys_global_out) {		\
																			\
	SortFunc<NumThreads, NumBits, ValueCount, UseScatterList,				\
		LoadFromTexture, EarlyExit>(										\
		keys_global_in, firstBlock, bucketCodes_global, bit,				\
		keys_global_out, 0, keys_global_out,								\
		0, 0, 0, 0, 0, 0,													\
		0, 0, 0, 0, 0, 0);													\
}




//GEN_SORT_FUNC(RadixSort_1, NUM_THREADS, 1, VALUE_COUNT, false,			\
//	LOAD_FROM_TEXTURE, false, NUM_BLOCKS)







/*
#define GEN_SORT_FUNC_

#ifdef VALUE_TYPE_NONE
void SORT_FUNC(const uint* keys_global_in, uint firstBlock,
	const uint* bucketCodes_global, uint bit, uint* keys_global_out) {

#elif defined(VALUE_TYPE_INDEX)
void SORT_FUNC(const uint* keys_global_in, uint firstBlock,
	const uint* bucketCodes_global, uint bit, uint* keys_global_out,
	uint* index_global_out) {

#elif defined(VALUE_TYPE_SINGLE)
void SORT_FUNC(const uint* keys_global_in, uint firstBlock,
	const uint* bucketCodes_global, uint bit, uint* keys_global_out, 
	const uint* value1_global_in, uint* value1_global_out) {

#elif defined(VALUE_TYPE_MULTI)
	// For VALUE_TYPE_MULTI, we have to pass each of the pointers in as 
	// individual arguments, not as arrays, or CUDA generates much worse code,
	// full of unified addressing instructions.
void SORT_FUNC(const uint* keys_global_in, uint firstBlock,
	const uint* bucketCodes_global, uint bit, uint* keys_global_out,
	uint numValueStreams,
//	const uint* values_global_in[6], uint* values_global_out[6]
	const uint* values1_global_in, const uint* values2_global_in,
	const uint* values3_global_in, const uint* values4_global_in,
	const uint* values5_global_in, const uint* values6_global_in,
	uint* values1_global_out, uint* values2_global_out,
	uint* values3_global_out, uint* values4_global_out,
	uint* values5_global_out, uint* values6_global_out) {
	*/