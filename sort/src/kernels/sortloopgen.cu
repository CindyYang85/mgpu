#include "hip/hip_runtime.h"
// One of these macros will be defined:
// VALUE_TYPE_NONE
// VALUE_TYPE_INDEX
// VALUE_TYPE_SINGLE
// VALUE_TYPE_MULTI

// If VALUE_TYPE_NONE is defined, we are working directly on keys. Otherwise, we
// produce fused keys and keep the key values in a register array.

#if defined(VALUE_TYPE_NONE)
	#define VALUE_COUNT 0
#elif defined(VALUE_TYPE_INDEX)
	#define VALUE_COUNT -1
#elif defined(VALUE_TYPE_SINGLE)
	#define VALUE_COUNT 1
#elif defined(VALUE_TYPE_MULTI)
	#define VALUE_COUNT 2
#endif

#define NUM_VALUES (VALUES_PER_THREAD * NUM_THREADS)

#ifdef BUILD_64

// 64bit kernels require more registers and so run less efficiently.
// To eliminate skills, give them so more regs. Note that this needs to be
// better optimized.

#ifdef VALUE_TYPE_NONE
#define REGS_PER_THREAD 32
#else
#define REGS_PER_THREAD 40
#endif


#else

#ifdef VALUE_TYPE_NONE
#define REGS_PER_THREAD 32
#else
#define REGS_PER_THREAD 36
#endif

#endif // BUILD_64


#define LOAD_FROM_TEXTURE true

texture<uint4, hipTextureType1D, hipReadModeElementType> keys_texture_in;

#include "sortloop.cu"

#define NUM_BLOCKS (32768 / ((~1 & (1 + REGS_PER_THREAD)) * NUM_THREADS))

GEN_SORT_LOOP(RadixSortLoop_1, NUM_THREADS, 1, VALUE_COUNT,					\
	LOAD_FROM_TEXTURE, NUM_BLOCKS)
GEN_SORT_LOOP(RadixSortLoop_2, NUM_THREADS, 2, VALUE_COUNT,					\
	LOAD_FROM_TEXTURE, NUM_BLOCKS)
GEN_SORT_LOOP(RadixSortLoop_3, NUM_THREADS, 3, VALUE_COUNT,					\
	LOAD_FROM_TEXTURE, NUM_BLOCKS)
GEN_SORT_LOOP(RadixSortLoop_4, NUM_THREADS, 4, VALUE_COUNT,					\
	LOAD_FROM_TEXTURE, NUM_BLOCKS)
GEN_SORT_LOOP(RadixSortLoop_5, NUM_THREADS, 5, VALUE_COUNT, 				\
	LOAD_FROM_TEXTURE, NUM_BLOCKS)
GEN_SORT_LOOP(RadixSortLoop_6, NUM_THREADS, 6, VALUE_COUNT,					\
	LOAD_FROM_TEXTURE, NUM_BLOCKS)
GEN_SORT_LOOP(RadixSortLoop_7, NUM_THREADS, 7, VALUE_COUNT, 				\
	LOAD_FROM_TEXTURE, NUM_BLOCKS)

