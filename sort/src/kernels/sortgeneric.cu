#include "hip/hip_runtime.h"
// Load 8 fused keys and produce a bucket total and a key offset within
// the thread-bucket. If 4 buckets are being counted, predInc is packed
// with bytes. If 8 buckets are being counted, predInc is packed with
// nibbles. In both cases, offsetsPacked is filled with bytes. This is
// because we'd have to expand the offsets to bytes anyway, so we do it
// here rather than first packing as nibbles.

// returns predInc packed into nibbles (for NUM_BITS=3) or bytes (NUM_BITS = 2).

DEVICE uint ComputeFusedKeyTotals(const Values fusedKeys, uint bit, 
	uint numBits, uint2& bucketsPacked, uint2& offsetsPacked) { 

	uint predInc = 0;
	const int BitsPerValue = (2 == numBits) ? 8 : 4;
		
	#pragma unroll
	for(int v = 0; v < 8; ++v) {
		// Note: if this is the first pass for a fused key sort, there is likely
		// optimization potential for exploiting knowledge that bit = 0.
		uint bucket = bfe(fusedKeys[v], bit, numBits);
		uint shift = BitsPerValue * bucket;

		// Insert the previous predInc to bucketsPacked.
		// Don't need to clear the high bits because bfi will do it
		uint prevPredInc = predInc>> shift;

		if(0 == v) {
			// set predInc with shift
			predInc = 1<< shift;
			offsetsPacked.x = 0;
			bucketsPacked.x = bucket;
		} else if(v < 4) {
			// bfi generates better code than shift and OR
			offsetsPacked.x = bfi(offsetsPacked.x, prevPredInc, 8 * v,
				BitsPerValue);
			bucketsPacked.x = bfi(bucketsPacked.x, bucket, 4 * v, 4);
		} else if(4 == v) {
			// If we're processing 3 bits we have to clear out the high bits of
			// prevPredInc, because otherwise they won't be overwritten to zero
			// by bfi.
			if(3 == numBits) prevPredInc &= 0x0f;
			offsetsPacked.y = prevPredInc;
			bucketsPacked.y = bucket;			
		} else {
			offsetsPacked.y = bfi(offsetsPacked.y, prevPredInc, 8 * (v - 4),
				BitsPerValue);
			bucketsPacked.y = bfi(bucketsPacked.y, bucket, 4 * (v - 4), 4);
		}

		if(v) predInc = shl_add(1, shift, predInc);
	}
	return predInc;
}


#include "sortscan1.cu"
#include "sortscan2.cu"
#include "sortscan3.cu"

// Read fused keys from shared memory, scan, and scatter the fused keys into
// strided shared memory. 

DEVICE void SortAndScatter(uint tid, Values fusedKeys, uint bit, uint numBits,
	bool loadKeysFromArray, uint* debug_global) {

	uint packed[4];

	if(loadKeysFromArray) {	
		volatile uint* threadData = scattergather_shared + 
			StridedThreadOrder(tid * VALUES_PER_THREAD);
			
		#pragma unroll
		for(int v = 0; v < 8; ++v)
			fusedKeys[v] = threadData[v];
	}
	
	if(1 == numBits) {
		SortScatter1(tid, fusedKeys, bit, packed, 0, 0, 0, debug_global);

	} else if(2 == numBits) {
		uint2 bucketsPacked;
		uint2 offsetsPacked;
		uint predInc = ComputeFusedKeyTotals(fusedKeys, bit, 2, bucketsPacked, 
			offsetsPacked);

		uint2 scanOffsets;
		scanOffsets = MultiScan2(tid, predInc, 0, 0, 0, debug_global);

		SortScatter2_8(scanOffsets, bucketsPacked, offsetsPacked, fusedKeys,
			packed, tid);

	} else if(3 == numBits) {
		uint2 bucketsPacked;
		uint2 offsetsPacked;
		uint predInc = ComputeFusedKeyTotals(fusedKeys, bit, 3, bucketsPacked, 
			offsetsPacked);

		uint4 scanOffsets;
		scanOffsets = MultiScan3(tid, Expand8Uint4To8Uint8(predInc),
			bucketsPacked, offsetsPacked, 0, 0, 0, debug_global);

		SortScatter3_8(scanOffsets, bucketsPacked, offsetsPacked, fusedKeys, 
			packed, tid);
	}
	__syncthreads();
}
	
