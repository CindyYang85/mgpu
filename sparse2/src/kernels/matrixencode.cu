#include "hip/hip_runtime.h"
#include "common.cu"


#ifdef MAT_TYPE_FLOAT
	typedef uint T;
#elif defined(MAT_TYPE_DOUBLE)
	typedef uint2 T;
#elif defined(MAT_TYPE_CFLOAT)
	typedef uint2 T;
#elif defined(MAT_TYPE_CDOUBLE)
	typedef uint4 T;
#endif



DEVICE2 void Zero(uint& x) { x = 0; }
DEVICE2 void Zero(uint2& x) { x = make_uint2(0, 0); }
DEVICE2 void Zero(uint4& x) { x = make_uint4(0, 0, 0, 0); }


// Flags to begin a segmented scan and to commit the scan to shared mem.
#define STORE_FLAG (1<< 25)

// Each thread initializes pointers dynamically.
struct Context {
	// Shared memory arrays. rowIndices and colIndices hold at least 
	int* rowIndices;
	int* colIndices;
	T* values;
	int* transposeBuffer;

	uint numValues;
	uint vt;
	int available;

	uint sharedScatter;
	uint sharedGather;

	// tid handles the value associated with (evalThread, evalValue) in the 
	// evaluation kernel.
	uint evalThread;
	
	// Help in converting strided index to thread index. If valuesPerThread is
	// odd, we can simply dereference into shared memory without any bank 
	// conflicts. If valuesPerThread is a power of 2, we use a stride of 33 
	// (add offset / WARP_SIZE). If valuesPerThread is even but not a power of
	// two, we'll have to eat a two-way serialization penalty, but can use the
	// native stride of 32.
	bool stridedIndex;
};

DEVICE int Index(const Context& context, int offset) {
	if(context.stridedIndex) offset += offset / WARP_SIZE;
	return offset;
}


DEVICE int TransposeLocal(int x, Context& context) {
	context.transposeBuffer[context.sharedScatter] = x;
	__syncthreads();
	x = context.transposeBuffer[context.sharedGather];
	__syncthreads();
	return x;
}

// Returns the number of values consumed.
DEVICE int ProcessRowIndices(uint tid, Context& context, 
	int*& colIndices_global, T*& values_global, int*& outputIndices_global) {

	volatile int* tempSpace_shared = (volatile int*)context.transposeBuffer;
	
	volatile int* first_shared = tempSpace_shared;
	volatile int* last_shared = tempSpace_shared + 32;
	volatile int* firstValueCache_shared = tempSpace_shared + 64;
	volatile int& lastTid_shared = tempSpace_shared[96];


	////////////////////////////////////////////////////////////////////////////
	// LOAD ROW INDICES AND FIND RANGE OF ACTUAL VALUES TO CONSUME IN THIS
	// BLOCK.
	// Values from rows firstRow through firstRow + 32 (exclusive) are
	// "available" and streamed to the encoded block. 

	// Clear the rowStartThread and rowEndThread arrays. Any index other than 
	// -1 indicates that the row in question is in the block.
	if(tid < 2 * WARP_SIZE)
		first_shared[tid] = -1;
	__syncthreads();

	// Get the first row, tid's row, and the row at tid + 1.
	int firstRow = context.rowIndices[0];
	int threadRow = context.rowIndices[tid];
	int nextRow = context.rowIndices[tid + 1];

	// endRow is one past the last row that can possibly be encountered in this
	// block.
	int endRow = firstRow + WARP_SIZE;

	// These values are true if the thread's value is in the block.
	int threadTest = (tid < context.available) && (threadRow < endRow);
	int nextTest = (tid + 1 < context.available) && (nextRow < endRow);

	// The last in-range thread writes to lastTid_shared.
	if(threadTest != nextTest) lastTid_shared = tid;
	__syncthreads();

	// Retrieve the tid of the last element in the block and its row index.
	uint lastTid = lastTid_shared;
	int lastRow = context.rowIndices[lastTid];

	// If threadRow or nextRow is out-of-bounds, set it to lastRow.
	int prevRow = context.rowIndices[tid - 1];
	prevRow = min(prevRow, lastRow);
	threadRow = min(threadRow, lastRow);
	nextRow = min(nextRow, lastRow);

	int rowDelta = threadRow - firstRow;

	// If this is the first value of this row in the block, store to
	// rowStart_shared.
	if(prevRow != threadRow) first_shared[rowDelta] = context.evalThread;
	if(threadRow != nextRow) last_shared[rowDelta] = context.evalThread;
	__syncthreads();


	////////////////////////////////////////////////////////////////////////////
	// CALCULATE SPECIAL OFFSETS FOR FIRST FOUR VALUES OF EACH EVAL THREAD.

	if(tid < WARP_SIZE) {
		// Load the index of the first row encontered in this thread.
		int offset1 = tid * context.vt;
		int row1 = context.rowIndices[Index(context, offset1)];
		if(offset1 > lastTid) row1 = lastRow;

		// Load the start thread for the first row encountered in this thread.
		int startRowTid = first_shared[row1];

		// Load the thread ranges for the tid'th row (rowDelta, not available
		// row). Find the number of store slots required for the tid'th row.
		int rowTid1 = first_shared[tid];
		int rowTid2 = last_shared[tid];
		bool rowValid = -1 != rowTid1;
		int rowSlotCount = rowTid1 ? (rowTid2 - rowTid1 + 1) : 0;

		// Scan the store slots for each row. The exclusive scan is stored in
		// shared memory. The inclusive scan is retained in the register x.
		tempSpace_shared[tid] = 0;
		volatile int* scan = tempSpace_shared + 16;
		scan[0] = rowSlotCount;
		int incScan = rowSlotCount;
		int excScan;
		#pragma unroll
		for(int i = 0; i < LOG_WARP_SIZE; ++i) {
			int offset = 1<< i;
			int y = scan[-offset];
			incScan += y;
			if(i < LOG_WARP_SIZE - 1) scan[0] = incScan;
			else {
				excScan = incScan - rowSlotCount;
				scan[0] = excScan;
			}
		}

		// THREAD INDEX A: thread scan offset.
		// Pull the scan offset of the first row encountered in this eval
		// thread. Offset by the distance between this eval thread and that
		// row's starting thread (startRowTid).
		int evalStoreOffset = tempSpace_shared[16 + row1] + tid - 
			startRowTid;

		// THREAD INDEX B, C: segmented scan distance.
		// Store a bit at the start of each available row in the scan array.
		// Read these back at tid and 32 + tid and use __ballot and __clz to
		// find the start of the sgement (that is, the most-significant set bit
		// at or before the current position).
		tempSpace_shared[tid] = 0;
		tempSpace_shared[32 + tid] = 0;
		if(rowValid) tempSpace_shared[excScan] = 1;

		uint scanStartX = __ballot(tempSpace_shared[tid]);
		uint scanStartY = __ballot(tempSpace_shared[WARP_SIZE + tid]);

		uint mask = 0xffffffff>> (31 - tid);
		uint distanceX = 31 - tid - __clz(mask & scanStartX);
		uint distanceY = scanStartY ? 
			(31 - tid - __clz(mask & scanStartY)) : 
			(63 - tid - __clz(scanStartX));

		// Count the total number of rows in the black.
		uint rowBits = __ballot(rowValid);
		uint precedingRows = __popc(bfi(0, 0xffffffff, 0, tid) & rowBits);
		uint numRows = __popc(rowBits);

		// THREAD INDEX D: last scan slot for each available row. 
		// If this is a valid row, store the last index for it. Otherwise, store
		// zero.
		uint lastRowSlot = rowValid ? (incScan - 1) : 0;
		uint target = rowValid ?
			precedingRows :
			(tid - precedingRows + numRows);

		// Store the four special terms.
		tempSpace_shared[tid] = evalStoreOffset<< 26;
		tempSpace_shared[32 + tid] = (distanceX<< 27) | ((tid < numRows)<< 26);
		tempSpace_shared[64 + tid] = distanceY<< 26;
		tempSpace_shared[96 + target] = lastRowSlot<< 26;


		// Store the global row indices.
	}
	__syncthreads();

	int storeBit = (threadRow != nextRow) ? STORE_FLAG : 0;

	////////////////////////////////////////////////////////////////////////////
	// LOAD COL INDICES AND ATTACH SPECIAL OFFSETS AND STORE FLAGS.

	uint colIndex = 0;
	if(tid < lastTid) colIndex = context.rowIndices[tid];
	if(threadRow != nextRow) colIndex |= STORE_FLAG;

	// Transpose to put the decorated column index into strided order, suitable
	// for storing to global memory.
	colIndex = TransposeLocal(colIndex, context);

	// Apply the special indices for the first four values for each eval thread.
	if(tid < 4 * WARP_SIZE) colIndex |= tempSpace_shared[tid];

	// Store the decorated column indices to global memory.
	*colIndices_global = colIndex;
	colIndices_global += context.numValues;

	
	////////////////////////////////////////////////////////////////////////////
	// LOAD VALUES AND TRANSPOSE AND STORE.
	
	T value;
	Zero(value);
	if(tid < lastTid) value = context.values[context.sharedGather];

	*values_global = value;
	values_global += context.numValues;

	return lastTid + 1;
}


DEVICE void MoveToFront(uint tid, int consumed, Context& context) {

	int index1 = Index(context, tid);
	int index2 = Index(context, context.numValues + tid);

	if(consumed < context.available) {
		// Move all the values left by consumed slots.
		int tidRow1 = context.rowIndices[tid];
		int tidCol1 = context.colIndices[index1];
		T val1 = context.values[index1];

		int tidRow2, tidCol2;
		T val2;
		if(tid < WARP_SIZE) {
			tidRow2 = context.rowIndices[context.numValues + tid];
			tidCol2 = context.colIndices[index2];
			val2 = context.values[index2];
		}
		__syncthreads();

		if(tid < WARP_SIZE) {
			int i = context.numValues + tid - consumed;
			int index = Index(context, i);			
			context.rowIndices[i] = tidRow2;
			context.colIndices[index] = tidCol2;
			context.values[index] = val2;
		}
		if(tid >= consumed) {
			int i = tid - consumed;
			int index = Index(context, i);
			context.rowIndices[i] = tidRow1;
			context.colIndices[index] = tidCol1;
			context.values[index] = val1;
		}
	}
	context.available -= consumed;
}

DEVICE void RepopulateSharedRows(uint tid, int numValues, int& remaining,
	const int*& rowIndices_global, const int*& colIndices_global,
	const T*& values_global, Context& context) {

	// Load up to numValues from rowIndices_global. Always load a multiple of
	// 32 values for coalescing.
	int remaining2 = ~(WARP_SIZE - 1) & context.available;
	int count = min(numValues - remaining2, remaining);

	if(tid < count) {
		int i = context.available + tid;
		int index = Index(context, i);
		context.rowIndices[i] = rowIndices_global[tid];
		context.colIndices[index] = colIndices_global[tid];
		context.values[index] = values_global[tid];
	}
	__syncthreads();
	
	rowIndices_global += count;
	colIndices_global += count;
	values_global += count;
	context.available += count;
}

template<int VT>
DEVICE2 void MatrixEncode(
	const int* rowIndices_global, const int* colIndices_global,
	const T* sparseValues_global, const int2* rangePairs_global,
	const int4* groupInfo_global, int* colIndicesOut_global,
	T* sparseValuesOut_global, int* rowIndicesOut_global,
	int* outputIndicesOut_global, int height) {

	const int Count = VT * WARP_SIZE;
	const int Stride = ((Count - 1) & Count) ? 32 : 33;

	// Keep the row indices with a stride of 32 for fast random access.
	__shared__ int rowIndices_shared[Count + WARP_SIZE];

	// Keep the col indices and values with a stride (33 for pow2 VT) for fast
	// transpose from strided order to thread order.
	__shared__ int colIndices_shared[Stride * (VT + 1)];
	__shared__ T values_shared[Stride * (VT + 1)];
	__shared__ int transpose_shared[Stride * VT];
	
	uint tid = threadIdx.x;	
	uint block = blockIdx.x;
	int2 rangePair = rangePairs_global[block];
	int remaining = rangePair.y - rangePair.x;

	Context context;
	context.available = 0;
	context.rowIndices = rowIndices_shared;
	context.colIndices = colIndices_shared;
	context.values = values_shared;
	context.transposeBuffer = transpose_shared;

	context.numValues = Count;
	context.vt = VT;

	// This should be the only divide in the kernel.
	context.evalThread = tid / VT;



	while(remaining) {
		RepopulateSharedRows(tid, Count, remaining, rowIndices_global, 
			colIndices_global, sparseValues_global, context);

		int consumed = ProcessRowIndices(tid, context, colIndicesOut_global,
			sparseValuesOut_global, outputIndicesOut_global);

		MoveToFront(tid, consumed, context);
	}

	// Cap out the last index.
}


#define DEFINE_ENCODE(vt)													\
extern "C" __global__ void MatrixEncode_##count(							\
	const int* rowIndices_global, const int* colIndices_global,				\
	const T* sparseValues_global, const int2* rangePairs_global,			\
	const int4* groupInfo_global, int* colIndicesOut_global,				\
	T* sparseValuesOut_global, int* rowIndicesOut_global,					\
	int* outputIndicesOut_global, int height) {								\
																			\
	MatrixEncode<vt>(rowIndices_global, colIndices_global,					\
		sparseValues_global, rangePairs_global, groupInfo_global,			\
		colIndicesOut_global, sparseValuesOut_global, rowIndicesOut_global, \
		outputIndicesOut_global, height);									\
}

DEFINE_ENCODE(4)


/*




template<int NUM_VALUES> 
ThreadContext<NUM_VALUES> BuildContext(uint tid, uint* shared) {

	// Include spacing in the three buffers to eliminate bank conflicts during
	// transpose.
	const int count = NUM_VALUES + (NUM_VALUES / WARP_SIZE) + WARP_SIZE;

	ThreadContext c;
	c.rowIndices = shared;
	c.colIndices = c.rowIndices + Count;
	c.values = (T*)(c.colIndices + Count);
	c.rowAvailability = (volatile uint*)(c.values + Count);
	c.lastThreadReduction = c.rowAvailability + WARP_SIZE;
	c.lastTidRow = c.lastThreadReduction + 2 * WARP_SIZE;

	c.lane = (WARP_SIZE - 1) & tid;
	c.warp = tid / WARP_SIZE;

	// Scatter/gather through shared memory to perform a conflict-free transpose
	// to put the data in thread order from strided order. This simplifies the
	// evaluation kernel.
	uint vt = NUM_VALUES / WARP_SIZE;
	uint sharedOffset = vt * c.lane;
	sharedOffset += sharedOffset / WARP_SIZE;
	c.sharedScatter = lane + warp * (WARP_SIZE + 1);
	c.sharedGather = sharedOffset + warp;

	// This division is a *slow* operation on GPU because there is no integer
	// division. However we only have to do this once per thread at the start
	// of the thread block. This value gets reused many times.
	c.evalThread = tid / vt;
	c.evalValue = tid - vt * c.evalThread;

	c.available = 0;
	c.threadCode = 0;
	if(0 == c.evalValue) c.threadCode |= FirstThreadRow;
	if(vt - 1 == c.evalValue) c.threadCode |= LastThreadRow;

	return c;
}


////////////////////////////////////////////////////////////////////////////////
// WriteThreadFlags

// Returns the last tid of the warp. 
DEVICE uint WriteThreadFlags(uint tid, ThreadContext context) {
	// These three row indices are enough to determine the head flags and if the
	// thread's value actually belongs in this block.
	int firstRow = context.rowIndices[0];
	int threadRow = context.rowIndices[tid];
	int subsequentRow = context.rowIndices[tid + 1];

	// endRow is one past the last row that can possibly be encountered in this
	// block.
	int endRow = firstRow + WARP_SIZE;

	// These values are true if the thread's value is in the block.
	int threadTest = (tid < context.available) && (threadRow < endRow);
	int subsequentTest = (tid + 1 < available) && (subsequentRow < endRow);

	// The last in-range thread writes to reduction[LAST_ROW_TID].
	if(threadTest != subsequentTest) *context.lastTidRow = tid;
	__syncthreads();

	// Retrieve the tid of the last element in the block and its row index.
	uint lastTid = *context.lastTidRow;
	int lastRow = context.rowIndices[lastTid];
	if(tid > lastTid) threadRow = lastRow;
	if(tid + 1 > lastTid) subsequentRow = lastRow;
	if(tid == context.numValues - 1) subsequentRow = 0x7fffffff;
	
	int precedingRow = context.rowIndices[max(tid,  1) - 1];
	if(tid > lastTid) precedingRow = lastRow;
	if(!tid) precedingRow = -1;

	int rowDelta = threadRow - firstRow;

	int threadFlags = 0;
	if(precedingRow < threadRow)
		threadFlags |= LastThreadRow;
	reductionCode |= 1<< 25;

	// Prepare the segmented scan codes for this value.
	uint threadFlags = context.threadCode;
	
	// If this value is in a different row from the preceding one, it STARTS a
	// segment.
	if(precedingRow < threadRow)
		threadFlags |= FirstThreadRow;

	// If this value is in a different row from the subsequent one, it ENDS a
	// segment.
	if(threadRow < subsequentRow) {
		threadFlags |= LastThreadRow;

		// Mark that a particular value has been encountered.
		// NOTE: Is this necessary?
		context.rowAvailability[rowDelta] = 1;
	}

	if(LastThreadRow & threadFlags) {
		uint code = rowDelta | (context.evalThread<< 20) | (1<< 19);
		context.lastThreadReduction[context.evalThread + rowDelta] = code;
	}

	__syncthreads();
	return lastTid;
}


////////////////////////////////////////////////////////////////////////////////

template<int NUM_VALUES>
DEVICE2 ComputeSegScan(uint tid, ThreadContext context) {
	

}



*/

