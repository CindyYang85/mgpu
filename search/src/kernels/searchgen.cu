#define DEVICE extern "C" __device__ __forceinline__
#define DEVICE2 __device__ __forceinline__

#define WARP_SIZE 32
#define LOG_WARP_SIZE 5

// Size of a memory segment
#define SEG_SIZE 128

#define SearchTypeLower 0
#define SearchTypeUpper 1
#define SearchTypeRange 2

// NOTE: SEG_LANES_32_BIT may be set to 32 bits for better performance if the
// alu:mem ratio can be brought down. SEG_LANES_64_BIT should not however be
// increased.
#define SEG_LANES_32_BIT 16
#define SEG_LANES_64_BIT 16

#define MAX_LEVELS 8


typedef unsigned int uint;
typedef __int64 int64;
typedef unsigned __int64 uint64;

#include <hip/device_functions.h>
#include <vector_functions.h>

// insert the first numBits of y into x starting at bit
DEVICE uint bfi(uint x, uint y, uint bit, uint numBits) {
	uint ret;
	asm("bfi.b32 %0, %1, %2, %3, %4;" : 
		"=r"(ret) : "r"(y), "r"(x), "r"(bit), "r"(numBits));
	return ret;
}


#include "buildtree.cu"

#include "searchtree.cu"
