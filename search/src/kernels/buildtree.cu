#include "hip/hip_runtime.h"
#define NUM_BUILD_THREADS 256
#define BUILD_BLOCKS_PER_SM 6

template<typename T> __forceinline__ __device__  
void BuildTree(const T* data, uint count, T* tree) {

	const int SegLanes = SEG_SIZE / sizeof(T);

	// Divide (rounding up) by SegLanes for the number of dest lanes.
	uint validDest = (count + SegLanes - 1) / SegLanes;

	// Round up to a multiple of the node size (SegLanes)
	uint numGid = ~(SegLanes - 1) & (validDest + SegLanes - 1);

	uint tid = threadIdx.x;
	uint block = blockIdx.x;
	uint gid = NUM_BUILD_THREADS * block + tid;

	if(gid < numGid) {
		uint index = SegLanes * gid + SegLanes - 1;
		index = min(index, count - 1);

		T x = data[index];
		tree[gid] = x;
	}
}


extern "C" __global__ __launch_bounds__(NUM_BUILD_THREADS, BUILD_BLOCKS_PER_SM)
void BuildTree4(const uint* data, uint count, uint* tree) {

	BuildTree(data, count, tree);
}

extern "C" __global__ __launch_bounds__(NUM_BUILD_THREADS, BUILD_BLOCKS_PER_SM)
void BuildTree8(const double* data, uint count,  double* tree) {

	BuildTree(data, count, tree);
}
