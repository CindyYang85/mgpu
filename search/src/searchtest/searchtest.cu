#include "hip/hip_runtime.h"
#include "../../../inc/mgpusearch.h"
#include "../../../util/cucpp.h"
#include <vector>
#include <algorithm>
#include <thrust/device_ptr.h>
#include <thrust/binary_search.h>

#ifdef _MSC_VER
#include <random>
#else
#include <tr1/random>
#endif

typedef __int64 int64;

// Define all the permutations for benchmarking. The actual iterations taken
// for each array size/query size pair is the product of their second counts.

const int NumArraySizes = 10;
const int ArraySizes[NumArraySizes][2] = {
	{ 50000, 3000 },		
	{ 100000, 1000 },	// 100 K
	{ 500000, 750 },
	{ 1000000, 500 },	// 1M
	{ 5000000, 200 },
	{ 10000000, 150 },	// 10M
	{ 20000000, 80 },	// 20M
	{ 40000000, 50 },	// 40M
	{ 60000000, 30 },	// 60M
	{ 80000000, 20 },	// 80M
};

const int NumQuerySizes = 5;
const int MaxQuerySize = 1000000;
const int QuerySizes[5][2] = {
	{ 1000, 10 },
	{ 10000, 3 },
	{ 100000, 1 },
};

const int NumTests = 4;


	
std::tr1::mt19937 mt19937;

template<typename T>
void FillVec(std::vector<T>& vec, int count) {
	std::tr1::uniform_int<T> r(0, 20);
	T cur = 0;
	vec.resize(count);
	for(int i(0); i < count; ++i) {
		vec[i] = cur;
		if(0 == r(mt19937)) ++cur;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Benchmark MGPU Search

template<typename T>
double MgpuBenchmark(searchEngine_t engine, int count, CuDeviceMem* values,
	searchType_t type, CuDeviceMem* btree, int numIterations, int numQueries,
	CuDeviceMem* keys, CuDeviceMem* indices, const T* valuesHost,
	const T* keysHost) {

	CuEventTimer timer;
	timer.Start();
	
	int size = (SEARCH_TYPE_INT32 == type) ? 4 : 8;
	int offset = 0;
	for(int it(0); it < numIterations; ++it) {
		offset += RoundUp(numQueries, 32);
		if(offset + numQueries > MaxQuerySize) offset = 0;

		searchStatus_t status = searchKeys(engine, count, type, 
			values->Handle(), SEARCH_ALGO_LOWER_BOUND,
			keys->Handle() + offset * size, numQueries, btree->Handle(),
			indices->Handle());
		if(SEARCH_STATUS_SUCCESS != status) {
			printf("FAIL!\n");
			exit(0);
		}
	}

	double elapsed = timer.Stop();
	double throughput = (double)numQueries * numIterations / elapsed;

	// Verify the results for the last set of queries run.
	std::vector<uint> results(numQueries);
	indices->ToHost(results);

	for(int i(0); i < numQueries; ++i) {
		const T* lower = std::lower_bound(valuesHost, valuesHost + count, 
			keysHost[offset + i]);
		if((lower - valuesHost) != results[i]) {
			printf("Failure in MGPU Search.\n");
			exit(0);
		}
	}

	return throughput;
}


////////////////////////////////////////////////////////////////////////////////
// Benchmark thrust


template<typename T>
double ThrustBenchmark(int count, CuDeviceMem* values, int numIterations,
	int numQueries,	CuDeviceMem* keys, CuDeviceMem* indices, 
	const T* valuesHost, const T* keysHost) {

	CuEventTimer timer;
	timer.Start();

	int offset = 0;
	for(int it(0); it < numIterations; ++it) {
		offset += RoundUp(numQueries, 32);
		if(offset + numQueries > MaxQuerySize) offset = 0;

		typedef thrust::device_ptr<T> P;
		P valuesP = P((T*)values->Handle());
		P keysP = P(((T*)keys->Handle()) + offset);
		
		thrust::device_ptr<uint> outputP((uint*)indices->Handle());
		thrust::lower_bound(
			valuesP, valuesP + count,
			keysP, keysP + numQueries,
			outputP);
	}

	double elapsed = timer.Stop();
	double throughput = (double)numQueries * numIterations / elapsed;

	// Verify the results for the last set of queries run.
	std::vector<uint> results(numQueries);
	indices->ToHost(results);

	for(int i(0); i < numQueries; ++i) {
		const T* lower = std::lower_bound(valuesHost, valuesHost + count, 
			keysHost[offset + i]);
		if((lower - valuesHost) != results[i]) {
			printf("Failure in thrust Search.\n");
			exit(0);
		}
	}

	return throughput;
}



////////////////////////////////////////////////////////////////////////////////
// Loop for running multiple tests over all query sizes (for the same sized
// data) for both MGPU and thrust.

template<typename T>
void BenchmarkLoop(CuContext* context, searchEngine_t engine, int count,
	int numIterations, double* mgpuBenchmarks, double* thrustBenchmarks) {

	////////////////////////////////////////////////////////////////////////////
	// Prepare the device assets.
	printf("ARRAY = %d elements.\n", count);

	// Fill the array with sorted values of random repetition.
	std::vector<T> values;
	FillVec(values, count);

	// Get a list of randomized keys for lower_bound search.
	std::vector<T> keys(MaxQuerySize);
	std::tr1::uniform_int<T> r(0, values.back());
	for(int i(0); i < MaxQuerySize; ++i)
		keys[i] = r(mt19937);

	// Move both arrays to device memory.
	DeviceMemPtr valuesDevice, keysDevice;
	hipError_t result = context->MemAlloc(values, &valuesDevice);
	result = context->MemAlloc(keys, &keysDevice);

	// Allocate result space.
	DeviceMemPtr indicesDevice;
	result = context->MemAlloc<uint>(MaxQuerySize, &indicesDevice);

	if(hipSuccess != result) {
		printf("Error allocating.\n");
		exit(0);
	}

	// Build the b-tree for MGPU Search.
	searchType_t type = (8 == sizeof(T)) ? SEARCH_TYPE_INT64 : 
		SEARCH_TYPE_INT32;

	int treeSize = searchTreeSize(count, type);
	DeviceMemPtr btreeDevice;
	result = context->ByteAlloc(treeSize, &btreeDevice);

	searchStatus_t status = searchBuildTree(engine, count, type, 
		valuesDevice->Handle(), btreeDevice->Handle());


	////////////////////////////////////////////////////////////////////////////
	

	for(int q(0); q < NumQuerySizes; ++q) {
		int querySize = QuerySizes[q][0];
		int iterations = numIterations * QuerySizes[q][1];

		mgpuBenchmarks[q] = 0;
		thrustBenchmarks[q] = 0;
		printf("\t(%d, %d):\n", count, querySize);

		for(int t(0); t < NumTests; ++t) {
			// Test MGPU
			double throughput = MgpuBenchmark(engine, count, valuesDevice,
				type, btreeDevice, iterations, querySize, keysDevice,
				indicesDevice, &values[0], &keys[0]);
			printf("\t\t%10.3f M/s\t\t", throughput / 1.0e6);
			mgpuBenchmarks[q] = std::max(mgpuBenchmarks[q], throughput);

			// Test thrust
			throughput = ThrustBenchmark(count, valuesDevice, iterations,
				querySize, keysDevice, indicesDevice, &values[0], &keys[0]);
			printf("\t\t%10.3f M/s\n", throughput / 1.0e6);
		}
	}
	printf("\n");
}


int main(int argc, char** argv) {

	hipInit(0);

	DevicePtr device;
	CreateCuDevice(0, &device);

	ContextPtr context;
	CreateCuContext(device, 0, &context);

	searchEngine_t engine = 0;
	searchStatus_t status = searchCreate("../../src/cubin/search.cubin",
		&engine);

	double mgpuThroughputs[2][NumArraySizes][NumQuerySizes];
	double thrustThroughputs[2][NumArraySizes][NumQuerySizes];

	for(int s(6); s < NumArraySizes; ++s) {

		// 32-bit
		printf("32-bit key search:\n");
		BenchmarkLoop<int>(context, engine, ArraySizes[s][0], ArraySizes[s][1],
			mgpuThroughputs[0][s], thrustThroughputs[0][s]);

		// 64-bit
		printf("64-bit key search:\n");
		BenchmarkLoop<int64>(context, engine, ArraySizes[s][0], 
			ArraySizes[s][1], mgpuThroughputs[1][s], thrustThroughputs[1][s]);
	}



	searchDestroy(engine);
}

/*

int main(int argc, char** argv) {

	typedef int64 T;
	searchType_t type = SEARCH_TYPE_INT64;

	hipInit(0);

	DevicePtr device;
	CreateCuDevice(0, &device);

	ContextPtr context;
	CreateCuContext(device, 0, &context);

	searchEngine_t engine = 0;
	searchStatus_t status = searchCreate("../../src/cubin/search.cubin",
		&engine);

	std::vector<T> values;
	FillVec(values, NumElements);

	DeviceMemPtr deviceData, deviceTree, deviceResults;
	context->MemAlloc(values, &deviceData);

	int treeSize = searchTreeSize(NumElements, type);

	context->ByteAlloc(treeSize, &deviceTree);

	status = searchBuildTree(engine, NumElements, type, 
		deviceData->Handle(), deviceTree->Handle());

	T last = values.back();

	// SEARCH
	const int NumQueries = 10000;
	T keys[NumQueries];
	for(int i(0); i < NumQueries; ++i) {
		float delta = (float)last / NumQueries;
		keys[i] = (int)(delta / 2 + i * delta);
	}
//	keys[NumQueries - 1] = 1000000;

	DeviceMemPtr keysDevice, indicesDevice;
	context->MemAlloc(keys, NumQueries, &keysDevice);
	context->MemAlloc<uint>(NumQueries, &indicesDevice);
	status = searchKeys(engine, NumElements, type, 
		deviceData->Handle(), SEARCH_ALGO_UPPER_BOUND, keysDevice->Handle(),
		NumQueries, deviceTree->Handle(), indicesDevice->Handle());

	std::vector<int> indicesHost;
	indicesDevice->ToHost(indicesHost);

	for(int i(0); i < NumQueries; ++i) {
		int j = indicesHost[i];
	//	printf("%I64d %d: (%I64d, %I64d)\n", keys[i], j, values[j - 1], values[j]);
		printf("%d\n", j);
	}

	searchDestroy(engine);
}*/

/*
struct BTreeCPU {
	int count;
	std::vector<int> data;	

	// Support up to 6 btree levels.
	int numLevels;
	int levelCounts[6];
	std::vector<int> levelData[6];
};

void CreateBTreeCPU(std::vector<int>& data, int count,
	std::auto_ptr<BTreeCPU>* ppTree) {

	std::auto_ptr<BTreeCPU> tree(new BTreeCPU);
	tree->numLevels = 0;
	tree->count = count;
	tree->data.swap(data);

	const int SEG_SIZE = 32;

	int level = 0;
	while(count > SEG_SIZE) {
		// Divide by 32 to get the size of the next btree level.
		int count2 = (count + SEG_SIZE - 1) / SEG_SIZE;

		// Round up to a multiple of 32 to make indexing simpler.
		int newCount = ~(SEG_SIZE - 1) & (count2 + SEG_SIZE - 1);
		tree->levelData[level].resize(newCount);

		// Prepare the subsampling.
		const int* source = level ? &tree->levelData[level - 1][0] :
			&tree->data[0];

		for(int i(0); i < newCount; ++i) {
			int j = std::min(SEG_SIZE * i + SEG_SIZE - 1, count - 1);
			tree->levelData[level][i] = source[j];
		}

		// Store the level count.
		tree->levelCounts[level++] = newCount;
		count = newCount;
	}
	tree->numLevels = level;

	// Swap the levels to put them in order.
	for(int i(0); i < level / 2; ++i) {
		tree->levelData[i].swap(tree->levelData[level - 1 - i]);
		std::swap(tree->levelCounts[i], tree->levelCounts[level - 1 - i]);
	}

	*ppTree = tree;
}*/









/*
searchStatus_t SEARCHAPI searchKeys(searchEngine_t engine, int count,
	searchType_t type, hipDeviceptr_t data, searchAlgo_t algo, hipDeviceptr_t keys, 
	int numQueries, hipDeviceptr_t tree, hipDeviceptr_t results) {
*/
/*
#include <cstdio>
#include <vector>
#include <memory>
#include <algorithm>
#include <random>

std::tr1::mt19937 mt19937;

// Build 

const int SEG_SIZE = 32;

struct BTree {
	int count;
	std::vector<int> data;	

	// Support up to 6 btree levels.
	int numLevels;
	int levelCounts[6];
	std::vector<int> levelData[6];
};

void CreateBTree(std::vector<int>& data, int count,
	std::auto_ptr<BTree>* ppTree) {

	std::auto_ptr<BTree> tree(new BTree);
	tree->numLevels = 0;
	tree->count = count;
	tree->data.swap(data);

	int level = 0;
	while(count > SEG_SIZE) {
		// Divide by 32 to get the size of the next btree level.
		int count2 = (count + SEG_SIZE - 1) / SEG_SIZE;

		// Round up to a multiple of 32 to make indexing simpler.
		int newCount = ~(SEG_SIZE - 1) & (count2 + SEG_SIZE - 1);
		tree->levelData[level].resize(newCount);

		// Prepare the subsampling.
		const int* source = level ? &tree->levelData[level - 1][0] :
			&tree->data[0];

		for(int i(0); i < newCount; ++i) {
			int j = std::min(SEG_SIZE * i + SEG_SIZE - 1, count - 1);
			tree->levelData[level][i] = source[j];
		}

		// Store the level count.
		tree->levelCounts[level++] = newCount;
		count = newCount;
	}
	tree->numLevels = level;

	// Swap the levels to put them in order.
	for(int i(0); i < level / 2; ++i) {
		tree->levelData[i].swap(tree->levelData[level - 1 - i]);
		std::swap(tree->levelCounts[i], tree->levelCounts[level - 1 - i]);
	}

	*ppTree = tree;
}

int GetOffset(int key, const int* node) {
	for(int i(0); i < SEG_SIZE; ++i)
		if(node[i] >= key) return i;
	return SEG_SIZE;
}
int GetOffset2(int key, const int* node, int offset, int count) {
	int end = std::min(offset + SEG_SIZE, count);
	for(int i(offset); i < end; ++i)
		if(node[i] >= key) return i;
	return end;
}

int lower_bound(const BTree& tree, int key) {
	int numLevels = tree.numLevels;
	int offset = 0;
	for(int level(0); level < numLevels; ++level) {
		int o2 = GetOffset(key, &tree.levelData[level][offset]);
		offset = SEG_SIZE * (offset + o2);
	}
	offset = GetOffset2(key, &tree.data[0], offset, tree.count);
	return offset;	
}

int main(int argc, char** argv) {
	const int NumElements = 20000;
	std::tr1::uniform_int<int> r(0, 32767);

	std::vector<int> data(NumElements);
	for(int i(0); i < NumElements; ++i)
		data[i] = r(mt19937);
	std::sort(data.begin(), data.end());

	std::auto_ptr<BTree> tree;
	CreateBTree(data, NumElements, &tree);

	int offset = lower_bound(*tree, 32700);

	return 0;
}
*/